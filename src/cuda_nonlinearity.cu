#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <complex.h>
#include <math.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#ifdef WITHPERF
#include "perflib.h"
#endif
#include "redef.h"
#include "cuda_overlap.h"
#include "cuda_kernels.h"

//#define MEASURE_EVENTS
#undef MEASURE_EVENTS
#ifdef MEASURE_EVENTS
#define EVENT_RECORD(ev,st) hipEventRecord(ev,st)
#else
#define EVENT_RECORD(ev,st)
#endif

/* some macros for the calculation of the linear index */
#define LIND3(x,y,z,ny,nz) x*ny*nz+y*nz+z
#define LIND4(x1,x2,x3,x4,nx2,nx3,nx4) x1*nx2*nx3*nx4+x2*nx3*nx4+x3*nx4+x4

extern _Bool dealiasing;
extern int li0,lj0,li0da,li1da,ly0da,nj0;
/*extern int lbida2, ubida2; */
extern double ve_max[2];

extern "C" void da_interpolate_to_fine_grid_x(double _Complex inarr[],double _Complex tmp_arr1[]);
extern void da_interpolate_to_fine_grid_x_orig(double _Complex inarr[],double _Complex tmp_arr1[]);
extern void da_filter_from_fine_grid_x(double _Complex nl_tmp1[], double _Complex nonlin[]);
extern void to_fourier_y_hp_only_on_device(double *inarr, hipfftDoubleComplex *outarr, int iStream);
//extern void to_real_y_hp_only_on_device(hipfftDoubleComplex *inarr, double *outarr, int howmany, int streamId);
extern void to_real_y_only_on_device(hipfftDoubleComplex *inarr, double *outarr, int howmany, int streamId);
/*extern "C" double sum_abs(double *array, int len);*/
extern "C" void c_transpose_cmplx(int n1, int n2, double _Complex *in_matrix,  
		       double _Complex *transposed_matrix);
extern "C" double c_maxval(double *,int);
extern "C" void initialize_fourier_cufft(int a_li0da, int a_ly0da);
extern "C" void finalize_fourier_cufft();
extern "C" long int cuda_fourier_get_memory_need_on_device(int a_li0da,int a_ly0da);

int lbg0;
double sum_abs(double *array, int len);
double sum_cabs(hipfftDoubleComplex *array,int len);
double sum_cabs(double _Complex *array,int len);
void show_xy_arr(double _Complex *array,int nrows, int ncols);
void show_xy_arr(hipfftDoubleComplex *array,int nrows, int ncols);
void error_handling_with_synchronize(const char *kernelname);
void cuda_OverlapTransferAndFFT(const double _Complex *hf_inarr, double *df_outarr, int dims[3]);
void cuda_OverlapFFTAndTransfer(double *dev_nonlin1_re, double _Complex *localrhs);

Streamdata *mystreams[nStreams];

hipfftDoubleComplex *dev_temparrblock, *dev_vexy, *dev_dgdxy, *dev_nonlin, *dev_localrhs;
//double _Complex *nonlin;
double *dev_pnl;
hipfftDoubleReal *dev_vexy_re, *dev_dgdxy_re, *dev_nonlin1_re;
unsigned long int allocatedDeviceMemory=0L;
static hipEvent_t start_event, end_event, event2;
#ifdef MEASURE_EVENTS
static hipEvent_t start_transfer[nStreams],end_transfer[nStreams],after_copy_and_zero[nStreams],after_transpose[nStreams],after_to_real[nStreams];
static float stime_transfer[nStreams],stime_copy_and_zero[nStreams],stime_transpose[nStreams],stime_to_real[nStreams];
#endif


extern "C" void cuda_initialize_nonlinearity_df(int a_lbg0,
						double *cptr_pnl_1d) {
  lbg0 = a_lbg0;
  hipMalloc((void**)&dev_pnl,li0da*sizeof(double));
  hipMemcpy(dev_pnl,cptr_pnl_1d,li0da*sizeof(double),hipMemcpyHostToDevice);
}

extern "C" void cuda_finalize_nonlinearity_df() {
  hipFree(dev_pnl);
}

/* we need some temporary arrays on the GPU, which are allocated
   in advance.
   hipfftDoubleComplex dev_cmplxblock[2*lbg0][lj0][li0];
   hipfftDoubleComplex dev_temparrblock[2*lbg0][li0da][nj0];
   hipfftDoubleReal dev_realblock[2*lbg0][li0da][ly0da];
*/

extern "C" int cuda_get_nearest_blocksize(int test_blocksize) {
  return (test_blocksize/nParts)*nParts;
}

extern "C" int cuda_get_device_count(void) {
  int number_of_devices;
  hipGetDeviceCount(&number_of_devices);
  return number_of_devices;
}

extern "C" void cuda_set_device(int device) {
  int number_of_devices;
  //long int free_mem, total_mem;

  hipGetDeviceCount(&number_of_devices);
  //printf("We have %u GPU devices.\n",number_of_devices);
  if (device>=number_of_devices) {
    device = 2;
  }
  hipSetDevice(device);

  //hipMemGetInfo((size_t*)&free_mem,(size_t*)&total_mem);
  /*printf("Total available memory on device %u is %lu, free is %lu.\n",
    device, total_mem, free_mem);*/
}
  
extern "C" void cuda_register_array(double _Complex **arr,int arr_size) {
  hipError_t cuda_err;
  //printf("registering arr at address %p, size=%u bytes.\n",(void*)arr,arr_size*sizeof(double _Complex));
  cuda_err = hipHostRegister((void*)arr,arr_size*sizeof(double _Complex),hipHostRegisterPortable);
  if (cuda_err!=hipSuccess) {
    printf("Error with hipHostRegister: %s\n",hipGetErrorString(cuda_err));
  }/* else {
    printf("Registered %lu bytes at address %p.\n",arr_size*sizeof(double _Complex),arr);
    }*/
}

extern "C" void cuda_unregister_array(double _Complex **arr) {
  hipError_t cuda_err;
  //printf("unregistering arr at address %p\n",(void*)arr);
  cuda_err = hipHostUnregister((void*)arr);
  if (cuda_err!=hipSuccess) {
    printf("Error with hipHostUnregister: %s\n",hipGetErrorString(cuda_err));
  }
}

extern "C" long int cuda_get_memory_need_on_device() {
  long int mem_need;
  //long int mem_sav;

  mem_need = 0L;
  /*mem_need = 2*2*lbg0*lj0*li0*sizeof(hipfftDoubleComplex);*/
  /* dev_vexy_re, dev_dgdxy_re */
  mem_need += 2*2*lbg0*ly0da*li0da*sizeof(hipfftDoubleReal);
  /* dev_nonlin1_re */
  mem_need += lbg0*ly0da*li0da*sizeof(hipfftDoubleReal);
  /* dev_temparrblock */
  mem_need += 2*lbg0*li0da*nj0*sizeof(hipfftDoubleComplex);
  /* dev_nonlin, dev_localrhs */
  mem_need += 2*lbg0*li0da*nj0*sizeof(hipfftDoubleComplex);

  /* in the overlap case, we have per stream the following memory needs */
  /* dp_data */
  mem_need += nStreams*(2*lbg0*li0*lj0)/nParts*sizeof(hipfftDoubleComplex);
  /* dp_temp */
  mem_need += nStreams*(2*lbg0*li0*(ly0da/2+1))/nParts*sizeof(hipfftDoubleComplex);
  /* dp_fordeal */
  mem_need += nStreams*(2*lbg0*li0*(ly0da/2+1))/nParts*sizeof(hipfftDoubleComplex);

  /*mem_sav = mem_need;
    printf("mem_need without fourier: %lu\n",mem_need);*/
  mem_need += cuda_fourier_get_memory_need_on_device(li0da,ly0da);
  /*printf("mem_need for fourier: %lu, all gathered : %lu\n",
    mem_need-mem_sav,mem_need);*/
  /* for the reduction for the maximum */
  mem_need += (2*NTILES*lbg0+2)*sizeof(hipfftDoubleReal);

  /* the prefactor for the nonlinearity */
  mem_need += li0da*sizeof(double);
  return mem_need;
}

extern "C" long int cuda_get_free_memory_on_device() {
  long int mem_free, mem_total;

  hipMemGetInfo((size_t*)&mem_free, (size_t*)&mem_total);
  return mem_free;
}

extern "C" void cuda_allocate_on_device(int device) {
  hipError_t cuda_err;
  int passed=1;
  int number_of_devices, thisDevice, iStream;
  struct hipDeviceProp_t deviceProperties;
  //long int mem_free,mem_free_start, mem_free_before_fourier;

  allocatedDeviceMemory = 0L;
  hipGetDeviceCount(&number_of_devices);
  //printf("We have %u GPU devices.\n",number_of_devices);
  if (device<number_of_devices) {
    hipSetDevice(device);
  } else {
    hipSetDevice(0);
  }
  hipGetDevice(&thisDevice);
  hipGetDeviceProperties(&deviceProperties,thisDevice);
  //printf("We are really using now device %u, name is %s.\n",thisDevice,deviceProperties.name);

  /*mem_free_start=cuda_get_free_memory_on_device();
    printf("Free device memory before allocating anything: %lu.\n",mem_free_start);*/
  /* we also need the real arrays */
  cuda_err = hipMalloc((void**)&dev_vexy_re,
			2*lbg0*ly0da*li0da*sizeof(hipfftDoubleReal));
  if (cuda_err!=hipSuccess) {
    printf("Error with memory allocation on the device for dev_vexy_re.\n");
    passed=0;
  } else {
    allocatedDeviceMemory += 2*lbg0*ly0da*li0da*sizeof(hipfftDoubleReal);
  }
  cuda_err = hipMalloc((void**)&dev_dgdxy_re,
			2*lbg0*ly0da*li0da*sizeof(hipfftDoubleReal));
  if (cuda_err!=hipSuccess) {
    printf("Error with memory allocation on the device for dev_dgdxy_re.\n");
    passed=0;
  } else {
    allocatedDeviceMemory += 2*lbg0*ly0da*li0da*sizeof(hipfftDoubleReal);
  }

  cuda_err = hipMalloc((void**)&dev_nonlin1_re,
			lbg0*ly0da*li0da*sizeof(hipfftDoubleReal));
  if (cuda_err!=hipSuccess) {
    printf("Error with memory allocation on the device for dev_nonlin1_re.\n");
    passed=0;
  } else {
    allocatedDeviceMemory += lbg0*ly0da*li0da*sizeof(hipfftDoubleReal);
  }

  cuda_err = hipMalloc((void**)&dev_temparrblock,
			2*lbg0*li0da*nj0*sizeof(hipfftDoubleComplex));
  if (cuda_err!=hipSuccess) {
    printf("Error with memory allocation on the device for dev_temparrblock.\n");
    passed=0;
  } else {
    allocatedDeviceMemory += 2*lbg0*li0da*nj0*sizeof(hipfftDoubleComplex);
  }

  cuda_err = hipMalloc((void**)&dev_nonlin,
			lbg0*li0da*nj0*sizeof(hipfftDoubleComplex));
  if (cuda_err!=hipSuccess) {
    printf("Error with memory allocation on the device for dev_nonlin.\n");
    passed=0;
  } else {
    allocatedDeviceMemory += lbg0*li0da*nj0*sizeof(hipfftDoubleComplex);
  }

  cuda_err = hipMalloc((void**)&dev_localrhs,
			lbg0*li0da*nj0*sizeof(hipfftDoubleComplex));
  if (cuda_err!=hipSuccess) {
    printf("Error with memory allocation on the device for dev_localrhs.\n");
    passed=0;
  } else {
    allocatedDeviceMemory += lbg0*li0da*nj0*sizeof(hipfftDoubleComplex);
  }
  
  /*printf("allocatedDeviceMemory so far is %lu\n",allocatedDeviceMemory);*/
  /*cuda_err = hipHostAlloc((void**)&nonlin,lbg0*li0da*nj0*sizeof(double _Complex),hipHostMallocDefault);
  if (cuda_err != hipSuccess) {
    printf("Error with pinned memory allocation on host for nonlin.\n");
    passed=0;
  }
  */
  if (!passed) printf("Error with allocation of device memory. Used %lu bytes.\n",allocatedDeviceMemory);

  /* Create all streams for later use of events and asynchronous execution. */
  for (iStream=0;iStream<nStreams;iStream++) {
    //printf("Creating Stream %u of %u....",iStream,nStreams);
    mystreams[iStream] = new Streamdata(li0,lj0,ly0da,2*lbg0);
    //printf("Successful.\n");
  }
  /*mem_free_before_fourier = cuda_get_free_memory_on_device();
  printf("Before fourier init: memory free = %lu. allocatedDeviceMemory = %lu, usedMemory = %lu\n",
  mem_free_before_fourier,allocatedDeviceMemory,mem_free_start-mem_free_before_fourier);*/
  initialize_fourier_cufft(li0da,ly0da);
  /*mem_free = cuda_get_free_memory_on_device();
  printf("After fourier init: memory free = %lu. \
allocatedDeviceMemory = %lu, \
usedMemory by fourier = %lu\n",
mem_free,allocatedDeviceMemory,mem_free_before_fourier-mem_free);*/
  //printf("End of cuda_allocate_on_device.\n");

  hipEventCreate(&start_event);
  hipEventCreate(&end_event);
  hipEventCreate(&event2);

#ifdef MEASURE_EVENTS
  for (iStream=0;iStream<nStreams;iStream++) {
    hipEventCreate(&(start_transfer[iStream]));
    hipEventCreate(&(end_transfer[iStream]));
    hipEventCreate(&(after_copy_and_zero[iStream]));
    hipEventCreate(&(after_transpose[iStream]));
    hipEventCreate(&(after_to_real[iStream]));
    stime_transfer[iStream]=0.0;
    stime_copy_and_zero[iStream]=0.0;
    stime_transpose[iStream]=0.0;
    stime_to_real[iStream]=0.0;
  }
#endif
}

extern "C" void cuda_free_on_device() {
  hipError_t cuda_err;
  int passed=1, iStream;

  cuda_err = hipFree(dev_vexy_re);
  if (cuda_err!=hipSuccess) {
    printf("Error with memory free on the device for dev_vexy_re.\n");
    passed=0;
  }
  cuda_err = hipFree(dev_dgdxy_re);
  if (cuda_err!=hipSuccess) {
    printf("Error with memory free on the device for dev_dgdxy_re.\n");
    passed=0;
  }
  cuda_err = hipFree(dev_nonlin1_re);
  if (cuda_err!=hipSuccess) {
    printf("Error with memory free on the device for dev_nonlin1_re.\n");
    passed=0;
  }

  cuda_err = hipFree(dev_temparrblock);
  if (cuda_err!=hipSuccess) {
    printf("Error with memory free on the device for dev_temparrblock.\n");
    passed=0;
  }

  cuda_err = hipFree(dev_nonlin);
  if (cuda_err!=hipSuccess) {
    printf("Error with memory free on the device for dev_nonlin.\n");
    passed=0;
  }

  cuda_err = hipFree(dev_localrhs);
  if (cuda_err!=hipSuccess) {
    printf("Error with memory free on the device for dev_nonlin.\n");
    passed=0;
  }

  /*cuda_err = hipHostFree(nonlin);
  if (cuda_err!=hipSuccess) {
    printf("Error with memory free on the host for nonlin.\n");
    passed=0;
  }
  */

  finalize_fourier_cufft();
  if (!passed) printf("NOT all device memory successfully freed.\n");

  for (iStream=0;iStream<nStreams;iStream++) {
    delete mystreams[iStream];
    //hipStreamDestroy(stream[iStream]);
  }
  hipEventDestroy(start_event);
  hipEventDestroy(event2);
  hipEventDestroy(end_event);

#ifdef MEASURE_EVENTS
  for (iStream=0;iStream<nStreams;iStream++) {
    hipEventDestroy(start_transfer[iStream]);
    hipEventDestroy(end_transfer[iStream]);
    hipEventDestroy(after_copy_and_zero[iStream]);
    hipEventDestroy(after_transpose[iStream]);
    hipEventDestroy(after_to_real[iStream]);
    printf("Stream %u\ntransfer: %f\ncopy_and_zero: %f\ntranspose: %f\nto_real: %f\n----------\n",iStream,
	 stime_transfer[iStream],stime_copy_and_zero[iStream], stime_transpose[iStream], stime_to_real[iStream]);
  }
#endif
}

#include "cuda_kernels.cu"
#include "reduction_kernel.cu"

/*  This routine is now wholly running on the GPU. We start with
    a memcpy of the necessary input parameters to the GPU and then
    do all (transpose,FFT,nonlin calculation, y-dealiasing, back-FFT)
    on the GPU and only back-memcpy at the end. */

extern "C" 
void cuda_calc_nonlinearity_df(const double _Complex *gy_chi, 
			       const double _Complex *g_block,
			       const double _Complex *vexy,
			       const double _Complex *dgdxy,
			       double _Complex *localrhs,
			       _Bool first) {
  /*double _Complex gy_chi[lbg0][lj0][li0], 
    double _Complex g_block[lbg0][lj0][lbi:ubi], 
    double _Complex vexy[lbg0][2][lj0][li0], 
    double _Complex dgdxy[lbg0][2][lj0][li0], 
    double _Complex localrhs[lbg0][lj0][li0]
  */
  /*Complex, Dimension(li1:li2,lj1:lj2,1:lbg0),Intent(inout) :: gy_chi
    Complex, Dimension(lbi:ubi,lj1:lj2,1:lbg0),Intent(in) :: g_block
    Complex, Dimension(li1:li2,lj1:lj2,2,1:lbg0),Intent(inout) :: vexy, dgdxy
    Complex, Dimension(li1:li2,lj1:lj2,1:lbg0),Intent(inout) :: localrhs  
    Logical, Intent(in):: first
  */

  /* Local variables */
  /*Real, Dimension(0:ly0da-1, 0:li0da/n_procs_y-1,2,1:lbg0) ::  vexy_re, dgdxy_re*/
  //double vexy_re[lbg0][2][li0da][ly0da], dgdxy_re[lbg0][2][li0da][ly0da];
#if 0
  double *vexy_re, *dgdxy_re;
  /* debugging */
  void *nonlin1_re;
  /* end debugging */
#endif
  /*Real, Dimension(0:ly0da-1, 0:li0da/n_procs_y-1,1:lbg0) ::  nonlin1_re, nonlin2_re, nonlin3_re*/
  //double nonlin1_re[lbg0][li0da][ly0da];
  /*Complex, Dimension(0:nj0-1, 0:li0da/n_procs_y-1) :: tmp_arr*/
  //double _Complex tmp_arr[lbg0][li0da][nj0];
  /*    Complex, Dimension(lbida2:ubida2,lj1:lj2,1:lbg0) :: nl_tmp1*/
  /*double _Complex nl_tmp1[lbg0][lj0][ubida2-lbida2+1];*/
  /*Complex, Dimension(li1:li2,lj1:lj2,1:lbg0) :: nonlin */
  //double _Complex nonlin[lbg0][lj0][li0];
  //double _Complex *nonlin;

  /* Local variables */
  int shared_mem;
  hipfftDoubleReal *dev_temp_max0, *dev_temp_max1,*dev_max_per_block;
  double temp_max;
  hipError_t cuda_err;
  dim3 grid, threadblock;
  int array_dims[3];
#if 0
  float time_of_calc_nonlin,time_of_mult_pre, time_of_pure_cuda;
#endif

  //hipEventRecord(start_event,0);
#if 0
  /* DEBUGGING THE parameters*/
  int full_blocksize = li0*lj0*lbg0;
  printf("-------------- START of cuda_calc_...:\n localrhs = %13.6e, vexy = %13.6e, dgdxy = %13.6e\n",
    sum_cabs((hipfftDoubleComplex*)localrhs,full_blocksize),
    sum_cabs((hipfftDoubleComplex*)vexy,2*full_blocksize),
    sum_cabs((hipfftDoubleComplex*)dgdxy,2*full_blocksize));
  /* end debug */
#endif
  /* ----------------------------------------
     Transfer to the GPU 
     ----------------------------------------*/
  //hipDeviceSynchronize();
  C_PERFON("deal_FFT",8);
  array_dims[0]=lj0;
  array_dims[1]=li0;
  array_dims[2]=2*lbg0;
  /*printf("array_dims = %u,%u,%u\n",array_dims[0],array_dims[1], array_dims[2]);*/
  cuda_OverlapTransferAndFFT(vexy,dev_vexy_re,array_dims);
  cuda_OverlapTransferAndFFT(dgdxy,dev_dgdxy_re,array_dims);
  /* We have to make sure, that the fft are completed and the arrays are available. */
  hipDeviceSynchronize();
  C_PERFOFF();
#if 0
  /* DEBUGGING */
  hipDeviceSynchronize();
  vexy_re = (double*)malloc(li0da*ly0da*2*lbg0*sizeof(double));
  dgdxy_re = (double*)malloc(li0da*ly0da*2*lbg0*sizeof(double));
  /* transfer back to CPU the transformed block */
  hipMemcpy(vexy_re,dev_vexy_re,
	     li0da*ly0da*2*lbg0*sizeof(double),
	     hipMemcpyDeviceToHost);
  hipMemcpy(dgdxy_re,dev_dgdxy_re,
	     li0da*ly0da*2*lbg0*sizeof(double),
	     hipMemcpyDeviceToHost);

  printf("\tAFTER FFT: vexy_re = %13.6e, dgdxy_re = %13.6e\n",
	 sum_abs((double *)vexy_re,2*li0da*ly0da*lbg0),
	 sum_abs((double *)dgdxy_re,2*li0da*ly0da*lbg0));

#if 0
  int i,j;
  printf("First part, first block:\n");
  for (i=0;i<li0da;i++) {
    for (j=0;j<ly0da;j++) {
      printf("%10.6f ",vexy_re[i*ly0da+j]);
    }
    printf("\n");
  }
  printf("\nSecond part, first block:\n");
  for (i=0;i<li0da;i++) {
    for (j=0;j<ly0da;j++) {
      printf("%10.6f ",vexy_re[li0da*ly0da+i*ly0da+j]);
    }
    printf("\n");
  }
  printf("\n");
#endif
  free(vexy_re);
  free(dgdxy_re);
  /* END DEBUGGING */
#endif

  // get max ExB velocity
  if (first) {
    /*ve_x_max_loc=maxval(vexy_re(:,:,2,:))
      ve_y_max_loc=maxval(vexy_re(:,:,1,:))
      ve_max(1)=max(ve_max(1),ve_x_max_loc)        
      ve_max(2)=max(ve_max(2),ve_y_max_loc)
    */
    C_PERFON("ve_max",6);
    hipMalloc((void**)&dev_max_per_block,2*NTILES*lbg0*sizeof(hipfftDoubleReal));
    hipMalloc((void**)&dev_temp_max0,sizeof(hipfftDoubleReal));
    hipMalloc((void**)&dev_temp_max1,sizeof(hipfftDoubleReal));

    /*grid.x=lbg0;
    threadblock.x=li0da;
    shared_mem = (li0da<64 ? 64 : li0da)*2*sizeof(double);*/
    /*printf("Calling maxval_per_block with grid(%u,%u) and block(%u,%u,%u), shared_mem = %u\n",
      grid.x,grid.y,threadblock.x,threadblock.y,threadblock.z,shared_mem);*/
    //cuda_maxval_per_block_old<<<grid,threadblock,shared_mem>>>(dev_vexy_re,dev_max_per_block,li0da,ly0da);

    grid.x=lbg0;
    grid.y=NTILES;
    threadblock.x=ly0da;
    threadblock.y=1;
    shared_mem = ((ly0da<32) ? 64 : 2*ly0da)*sizeof(double);
    cuda_maxval_per_block<<<grid,threadblock,shared_mem>>>(dev_vexy_re,dev_max_per_block,li0da,ly0da);
    error_handling_with_synchronize("cuda_maxval_per_block");
    /* now we have the maxima for each block in dev_max_per_block.
       This has to be further reduced. */

    /* Debug output */
    /*
    double *max_per_block ,max0,max1;
    max_per_block =(double*)malloc(2*NTILES*lbg0*sizeof(double));
    hipMemcpy(max_per_block,dev_max_per_block,2*NTILES*lbg0*sizeof(double),hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("First part:\n");
    max0 = 0.0;
    for (i=0;i<NTILES*lbg0;i++) {
      printf("%10.6f ",max_per_block[i]);
      max0 = (max_per_block[i]>max0) ? max_per_block[i] : max0;
    }
    //printf("\n -- max0 = %f\nSecond part:\n",max0);
    printf("\nSecond part:\n");
    max1=0.0;
    for (i=0;i<NTILES*lbg0;i++) {
      printf("%10.6f ",max_per_block[NTILES*lbg0+i]);
      max1 = (max_per_block[NTILES*lbg0+i]>max1) ? max_per_block[NTILES*lbg0+i] : max1;
    }
    //printf("\n -- max1 = %f\n\n",max1);
    printf("\n");
    free(max_per_block);
    */
    /* end of Debug output */

    /* Further reduction */
    reduce<double>(NTILES*lbg0,6,dev_max_per_block,dev_temp_max0,REDUCTION_MAX,0);
    error_handling_with_synchronize("reduce<double>0");

    reduce<double>(NTILES*lbg0,6,dev_max_per_block,dev_temp_max1,REDUCTION_MAX,NTILES*lbg0);
    error_handling_with_synchronize("reduce<double>1");


    /* Freeing of dev_temp_max! */
    /* copying to the host */
    cuda_err = hipMemcpy(&temp_max,dev_temp_max0,sizeof(double),hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    if (cuda_err!=hipSuccess) {
      printf("Error with Memcpy. (dev_temp_max0 -> temp_max)\n");
      printf("%s\n",hipGetErrorString(cuda_err));
    } 

    //printf("temp_max0 = %f, ve_max[1] = %f\n",temp_max,ve_max[1]);
    /* max of vexy_re(0) is y component, max of vexy_re(1) is x component */
    ve_max[1] = fmax(temp_max,ve_max[1]);


    cuda_err = hipMemcpy(&temp_max,dev_temp_max1,sizeof(double),hipMemcpyDeviceToHost);
    //hipDeviceSynchronize();
    if (cuda_err!=hipSuccess) {
      printf("Error with Memcpy. (dev_temp_max1 -> temp_max)\n");
      printf("%s\n",hipGetErrorString(cuda_err));
    } 
    
    //printf("temp_max1 = %f, ve_max[0] = %f\n",temp_max,ve_max[0]);
    ve_max[0] = fmax(temp_max,ve_max[0]);

    hipFree(dev_max_per_block);
    hipFree(dev_temp_max0);
    hipFree(dev_temp_max1);

    //printf("ve_max = %f %f \n",ve_max[0],ve_max[1]);
    C_PERFOFF();
  }

  /* make sure, that all streams are synchronized with the host */
  //hipDeviceSynchronize();
  // compute the 'standard' nonlinear term, which is also used in the Arakawa representation
  /*nonlin1_re = -vexy_re(:,:,1,:)*dgdxy_re(:,:,2,:) + vexy_re(:,:,2,:)*dgdxy_re(:,:,1,:);
   There is no array syntax in C, so we have either to use explicit index syntax or use
   a BLAS function.*/


  /* While the nonlinearity is calculated, we copy the old localrhs to the GPU. */
  hipMemcpyAsync(dev_localrhs,localrhs,li0*lj0*lbg0*sizeof(double _Complex),hipMemcpyHostToDevice,mystreams[0]->cudaStream);

  C_PERFON("nonlin",6);
  C_PERFON("nl_kernl",8);
  /* We call the comp_stand_nonlin kernel. */
  comp_stand_nonlin_wrapper(dev_vexy_re,dev_dgdxy_re,dev_nonlin1_re,li0da,ly0da,nStreams-1);
  hipDeviceSynchronize();
  C_PERFOFF();

#if 0
  /* debug */
  nonlin1_re = malloc(li0da*ly0da*lbg0*sizeof(double));
  /* transfer back to CPU the transformed block */
  hipMemcpy(nonlin1_re,dev_nonlin1_re,
	     li0da*ly0da*lbg0*sizeof(double),
	     hipMemcpyDeviceToHost);
  printf("nonlin1_re = %15.6e \n",sum_abs((double*)nonlin1_re,li0da*ly0da*lbg0));
  free(nonlin1_re);
  /* end debug */
#endif

  //C_PERFON("ovrlp_2",7);
  cuda_OverlapFFTAndTransfer(dev_nonlin1_re,localrhs);
  hipDeviceSynchronize();
  //C_PERFOFF();
  C_PERFOFF();
  //hipEventRecord(event2,0);
  /*hipDeviceSynchronize();

  int lij0 = li0*lj0;
  for (klmn=0;klmn<lbg0;klmn++) {
    for (j=0;j<lj0;j++) {
      for (i=0;i<li0;i++) {
	//localrhs[klmn*lij0+j*li0+i] += cptr_pnl_1d[i]*nonlin[klmn*lij0+j*li0+i];
	localrhs[klmn*lij0+j*li0+i] += nonlin[klmn*lij0+j*li0+i];
      }
    }
  }
  */
  /*printf("after addition: localrhs = %f\n",
    sum_cabs((double _Complex *)localrhs,lbg0*li0*lj0));*/
  //hipEventRecord(end_event,0);
#if 0
  hipEventSynchronize(end_event);
  hipEventElapsedTime(&time_of_calc_nonlin,start_event,end_event);
  hipEventElapsedTime(&time_of_mult_pre,event2,end_event);
  hipEventElapsedTime(&time_of_pure_cuda,start_event,event2);
  printf("cuda nonlin time = %f ms, t(mult_pre) = %f ms, t(pure_cuda) = %f ms\n",time_of_calc_nonlin,time_of_mult_pre,time_of_pure_cuda);
#endif
}

void cuda_OverlapFFTAndTransfer(double *dev_nonlin1_re, double _Complex *localrhs) {
  int iStream, iPart;
  int nXYPlanesPerPart, dimXYPlane;
  dim3 grid, threadblock;
#if 0
  double _Complex *tmp_arr;
  int j,klmn;
#endif

  nXYPlanesPerPart = lbg0/nParts;
  dimXYPlane = li0da*ly0da;

  iStream=0;
  for (iPart=0;iPart<nParts;iPart++) {

    to_fourier_y_hp_only_on_device(dev_nonlin1_re+iPart*nXYPlanesPerPart*dimXYPlane,
				   (hipfftDoubleComplex*)(mystreams[iStream]->dp_temp),iStream);

#if 0
    hipDeviceSynchronize();
    tmp_arr = (double _Complex*)malloc(li0da*nj0*nXYPlanesPerPart*sizeof(hipfftDoubleComplex));
    hipMemcpy(tmp_arr,mystreams[iStream]->dp_temp,
	li0da*nj0*nXYPlanesPerPart*sizeof(double _Complex),
	hipMemcpyDeviceToHost);

    for (klmn=0;klmn<nXYPlanesPerPart;klmn++) {
      printf("klmn=%u, %f\n",klmn, sum_cabs(tmp_arr+klmn*li0da*nj0,li0da*nj0));
    }
    /*for (j=0;j<nj0;j++) {
      printf("(%f %f) ",creal(tmp_arr[j]),cimag(tmp_arr[j]));
      }
      printf("\n");*/
    
    printf("tmp_arr after to_fourier_y_hp is %f\n",sum_cabs(tmp_arr,
							    li0da*nj0*nXYPlanesPerPart));
    free(tmp_arr);
#endif
    
    // Transpose and remove zeros in y for dealiasing
    transpose_wrapper((hipfftDoubleComplex*)(mystreams[iStream]->dp_data),
		      (hipfftDoubleComplex*)(mystreams[iStream]->dp_temp), nj0,li0da, 
		      nXYPlanesPerPart,iStream);

#if 0
    /* Begin DEBUG */
    hipDeviceSynchronize();
    tmp_arr = (double _Complex*)malloc(li0da*nj0*nXYPlanesPerPart*sizeof(hipfftDoubleComplex));
    hipMemcpy(tmp_arr,mystreams[iStream]->dp_data,
	li0da*nj0*nXYPlanesPerPart*sizeof(double _Complex),
	hipMemcpyDeviceToHost);
    
    printf("after backtranspose, dp_data is %f\n",sum_cabs(tmp_arr,
							   li0da*nj0*nXYPlanesPerPart));
    free(tmp_arr);
    /* End DEBUG */
#endif

    /* The transposed nonlinearity is now in mystreams[iStream]->dp_data. We multiply it with 
       the prefactor dev_pnl. */
    grid.x=nXYPlanesPerPart;
    grid.y = nj0; grid.z=1;
    threadblock.x=li0da; threadblock.y=1; threadblock.z=1;
    copy_with_pnl<<<grid,threadblock,0,mystreams[iStream]->cudaStream>>>
      ((hipfftDoubleComplex*)mystreams[iStream]->dp_data,
       (hipfftDoubleComplex*)dev_localrhs+iPart*nXYPlanesPerPart*li0da*nj0,
       /*(hipfftDoubleComplex*)mystreams[iStream]->dp_temp,*/
       (hipfftDoubleReal*)dev_pnl,li0da,nj0);
    //hipStreamSynchronize(mystreams[iStream]->cudaStream);
#if 0
    /* Begin DEBUG */
    hipDeviceSynchronize();
    tmp_arr = (double _Complex*)malloc(li0da*nj0*nXYPlanesPerPart*sizeof(hipfftDoubleComplex));
    hipMemcpy(tmp_arr,
	       (hipfftDoubleComplex*)dev_localrhs+iPart*nXYPlanesPerPart*li0da*nj0,
	       /*mystreams[iStream]->dp_temp,*/
	       li0da*nj0*nXYPlanesPerPart*sizeof(double _Complex),
	       hipMemcpyDeviceToHost);
    
    printf("after copy_with_pnl, dev_localrhs is %f\n",sum_cabs(tmp_arr,
								li0da*nj0*nXYPlanesPerPart));
    free(tmp_arr);
    /* End DEBUG */
#endif


    hipMemcpyAsync(localrhs + iPart*nXYPlanesPerPart*li0da*nj0,
		    dev_localrhs + iPart*nXYPlanesPerPart*li0da*nj0,
		    /*mystreams[iStream]->dp_temp,*/
		    li0da*nj0*nXYPlanesPerPart*sizeof(double _Complex),
		    hipMemcpyDeviceToHost,mystreams[iStream]->cudaStream);
    /*hipMemcpy(nonlin,dev_nonlin,
      li0da*nj0*lbg0*sizeof(double _Complex),
      hipMemcpyDeviceToHost);*/
    iStream = (++iStream)%nStreams;
  }
  //hipDeviceSynchronize();
#if 0
  hipDeviceSynchronize();
  printf("localrhs = %15.6e\n",sum_cabs((hipfftDoubleComplex*)localrhs,lbg0*li0da*nj0));
#endif
}


void cuda_OverlapTransferAndFFT(const double _Complex *hf_inarr, double *df_outarr, int dims[3]) {

  int dimXYPlane=dims[0]*dims[1];
  int nXYPlanesPerPart, iStream,iPart;
  dim3 grid,threadblock;
  hipError_t cuda_err;
#ifdef MEASURE_EVENTS
  int nextStream;
  float time_transfer,time_copy_and_zero,time_transpose,time_to_real;
#endif

#if 0
  void *hp_temp;
  int klmn, offset,i,j;
#endif
  
  //printf("Starting cuda_OverlapTransferAndFFT.\n");
  if (dims[2]%nParts != 0) {
    printf("dims[2]=%u not divisable by nParts = %u\n",dims[2],nParts);
    exit(1);
  }
  nXYPlanesPerPart = dims[2]/nParts;
  /*printf("Working on %u parts, total number of xy-planes %u => each part contains %u XY-planes.\n \
    nStreams = %u\n",
    nParts,dims[2],nXYPlanesPerPart,nStreams);*/

  iStream = 0;
  for (iPart=0;iPart<nParts;iPart++) {
    /* copy a part to the device in stream iStream */
    EVENT_RECORD(start_transfer[iStream],mystreams[iStream]->cudaStream);
    hipMemcpyAsync(mystreams[iStream]->dp_data,&hf_inarr[iPart*nXYPlanesPerPart*dimXYPlane],
		    nXYPlanesPerPart*dimXYPlane*sizeof(double _Complex),
		    hipMemcpyHostToDevice,mystreams[iStream]->cudaStream);
    EVENT_RECORD(end_transfer[iStream],mystreams[iStream]->cudaStream);
    /*cuda_err = hipStreamSynchronize(mystreams[iStream]->cudaStream);
      if (cuda_err != hipSuccess) printf("overlap Memcpy: %s\n",hipGetErrorString(cuda_err));*/

#if 0
    /* debug */
    printf("sum_cabs(hp_inarr) = %f, dimXYPlane = %u, element = %f+%fi\n",
	   sum_cabs((hipfftDoubleComplex*)&hf_inarr[iPart*nXYPlanesPerPart*dimXYPlane],
		    nXYPlanesPerPart*dimXYPlane), dimXYPlane, creal(hf_inarr[0]),cimag(hf_inarr[0]));
    /* end debug */
#endif

    /* instead of first transposing and then afterwards do the copy and zero for dealiasing in y,
       we do the copy and zero first for better memory coalescing. 
       dims[0]=lj0, dims[1]=li0 */

    copy_and_zero_for_dealiasing_wrapper((hipfftDoubleComplex*)mystreams[iStream]->dp_data,
					 dims[1],dims[0],nXYPlanesPerPart,
					 (hipfftDoubleComplex*)mystreams[iStream]->dp_fordeal,
					 dims[1],ly0da/2+1,iStream);

    /*grid.x=ly0da/2+1;         grid.y=nXYPlanesPerPart;
    threadblock.x=dims[1];    threadblock.y=1;

    dev_copy_and_zero_for_dealiasing_new<<<grid,threadblock,0,mystreams[iStream]->cudaStream>>>
      ((hipfftDoubleComplex*)mystreams[iStream]->dp_data,dims[1],dims[0],
       (hipfftDoubleComplex*)mystreams[iStream]->dp_fordeal,dims[1],ly0da/2+1);
    */
    //cuda_err = hipStreamSynchronize(mystreams[iStream]->cudaStream);
    cuda_err = hipGetLastError();
    if (cuda_err != hipSuccess) printf("%u: after dev_copy_and_zero_for_dealiasing_new: %s\n",
					iStream,hipGetErrorString(cuda_err));
    EVENT_RECORD(after_copy_and_zero[iStream],mystreams[iStream]->cudaStream);

#if 0
    /* debug */
    hp_temp = malloc(nXYPlanesPerPart*dims[1]*(ly0da/2+1)*sizeof(double _Complex));
    hipMemcpy(hp_temp,mystreams[iStream]->dp_fordeal,
	       nXYPlanesPerPart*dims[1]*(ly0da/2+1)*sizeof(double _Complex),
	       hipMemcpyDeviceToHost);
    printf("total sum_cabs(dp_fordeal[%u]) = %f\n",iStream,
	   sum_cabs((hipfftDoubleComplex*)hp_temp,nXYPlanesPerPart*dims[1]*(ly0da/2+1)));
    free(hp_temp);
    /* end debug */
#endif

    /* compute FFT in iStream */
    transpose_wrapper((hipfftDoubleComplex*)mystreams[iStream]->dp_temp, 
		      (hipfftDoubleComplex*)mystreams[iStream]->dp_fordeal, 
		      dims[1],ly0da/2+1,nXYPlanesPerPart,iStream);
    EVENT_RECORD(after_transpose[iStream],mystreams[iStream]->cudaStream);

#if 0
    /* debug */
    hp_temp = malloc(nXYPlanesPerPart*dimXYPlane*sizeof(double _Complex));
    hipMemcpy(hp_temp,mystreams[iStream]->dp_temp,nXYPlanesPerPart*dimXYPlane*sizeof(double _Complex),
	       hipMemcpyDeviceToHost);
    printf("total sum_abs(dp_temp[%u]) = %f\n",iStream,
	   sum_cabs((double _Complex*)hp_temp,nXYPlanesPerPart*dimXYPlane));

    klmn= 0;
    offset = klmn*dimXYPlane;
    for (i=0;i<dims[1];i++) {
      for (j=0;j<dims[0];j++) {
	printf("(%10.3e %10.3e) ",creal(((double _Complex*)hp_temp)[offset+i*dims[0]+j]),
	       cimag(((double _Complex*)hp_temp)[offset+i*dims[0]+j]));
      }
      printf("\n");
    }
    printf("\n");

    /*for (klmn=0;klmn<2*lbg0;klmn++) {
      printf("klmn = %u, sum_cabs(hp_temp) = %f\n",klmn,
	     sum_cabs(((double _Complex*)hp_temp)+klmn*dimXYPlane,dimXYPlane));
	     }*/
    free(hp_temp);
    /* end debug */
#endif

    to_real_y_only_on_device((hipfftDoubleComplex*)mystreams[iStream]->dp_temp,
			     df_outarr+(iPart*nXYPlanesPerPart*li0da*ly0da),
			     nXYPlanesPerPart,iStream);
    EVENT_RECORD(after_to_real[iStream],mystreams[iStream]->cudaStream);

#if 0
    /* debug */
    hp_temp = malloc(nXYPlanesPerPart*li0da*ly0da*sizeof(double));
    hipMemcpy(hp_temp,df_outarr+(iPart*nXYPlanesPerPart*li0da*ly0da),
	       nXYPlanesPerPart*li0da*ly0da*sizeof(double),
	       hipMemcpyDeviceToHost);
    /*for (klmn=0;klmn<nXYPlanesPerPart;klmn++) {
      printf("klmn = %u, %u, sum_cabs(df_outarr) = %17.6e\n",klmn,klmn*li0da*ly0da,
	     sum_abs(((double*)hp_temp)+klmn*li0da*ly0da,li0da*ly0da));
	     }*/

    printf("total sum_abs(df_outarr) = %f\n",sum_abs((double*)hp_temp,nXYPlanesPerPart*li0da*ly0da));
    free(hp_temp);
    /* end debug */
#endif

    /* stream iStream is now used as next, hence we have to 
       get the events before restart of the stream. */
#ifdef MEASURE_EVENTS
    nextStream = (iStream+1)%nStreams;
    /*
    if (hipStreamWaitEvent(mystreams[iStream]->cudaStream,after_to_real[iStream],0)==hipSuccess) {
      cuda_err = hipEventElapsedTime(&time_transfer,start_transfer[iStream],end_transfer[iStream]);
      if (cuda_err==hipSuccess) stime_transfer[iStream] += time_transfer;
      cuda_err = hipEventElapsedTime(&time_copy_and_zero,end_transfer[iStream],after_copy_and_zero[iStream]);
      if (cuda_err==hipSuccess) stime_copy_and_zero[iStream] += time_copy_and_zero;
      cuda_err = hipEventElapsedTime(&time_transpose,after_copy_and_zero[iStream],after_transpose[iStream]);
      if (cuda_err==hipSuccess) stime_transpose[iStream] += time_transpose;
      cuda_err = hipEventElapsedTime(&time_to_real,after_transpose[iStream],after_to_real[iStream]);
      if (cuda_err==hipSuccess) stime_to_real[iStream] += time_to_real;
    */
    if (hipEventQuery(after_to_real[nextStream])==hipErrorNotReady) {
      cuda_err = hipEventSynchronize(after_to_real[nextStream]);
    }
    cuda_err = hipEventElapsedTime(&time_transfer,start_transfer[nextStream],end_transfer[nextStream]);
    if (cuda_err==hipSuccess) stime_transfer[nextStream] += time_transfer;
    cuda_err = hipEventElapsedTime(&time_copy_and_zero,end_transfer[nextStream],after_copy_and_zero[nextStream]);
    if (cuda_err==hipSuccess) stime_copy_and_zero[nextStream] += time_copy_and_zero;
    cuda_err = hipEventElapsedTime(&time_transpose,after_copy_and_zero[nextStream],after_transpose[nextStream]);
    if (cuda_err==hipSuccess) stime_transpose[nextStream] += time_transpose;
    cuda_err = hipEventElapsedTime(&time_to_real,after_transpose[nextStream],after_to_real[nextStream]);
    if (cuda_err==hipSuccess) stime_to_real[nextStream] += time_to_real;
    /* delete the errors. We get an error for the first call of the hipEventElapsedTime as the
       events used there have not been called before. This gives an Invalid Resource Handle error.
       To remove it from the error stack, we have to call hipGetLastError(). */
    
    hipGetLastError();

#endif
    /* increment the stream id for the next loop cycle */
    iStream = (++iStream)%nStreams;
  }
}

double sum_cabs(double _Complex *array,int len) {
  return sum_cabs((hipfftDoubleComplex*)array, len);
}

double sum_cabs(hipfftDoubleComplex *array,int len) {
  double sum;
  int i;

  sum = 0.0;
  for (i=0;i<len;i++) {
    sum += sqrt(array[i].x*array[i].x+array[i].y*array[i].y);
  }
  return sum;
}

void show_xy_arr(hipfftDoubleComplex *array,int nrows, int ncols) {
  int i,j,lind;
  for (i=0;i<nrows;i++) {
    for (j=0;j<ncols;j++) {
      lind = i*ncols+j;
      printf("(%5.2f %5.2f)",array[lind].x, array[lind].y);
    }
    printf("\n");
  }
}

void show_xy_arr(double _Complex *array,int nrows, int ncols) {
  int i,j,lind;
  for (i=0;i<nrows;i++) {
    for (j=0;j<ncols;j++) {
      lind = i*ncols+j;
      printf("(%5.2f %5.2f)",creal(array[lind]),cimag(array[lind]));
    }
    printf("\n");
  }
}

void error_handling_with_synchronize(const char *kernelname) {
  hipError_t cuda_err;

  cuda_err = hipGetLastError();
  if (cuda_err != hipSuccess) {
    printf("Error with kernel launch.(%s)\n",kernelname);
    printf("%s\n",hipGetErrorString(cuda_err));
  }
  cuda_err = hipDeviceSynchronize();
  if (cuda_err != hipSuccess) {
    printf("Error with Synchronize after kernel launch. (%s)\n",kernelname);
    printf("%s\n",hipGetErrorString(cuda_err));
  }
}

/* calculate the sum of the absolute values of an array */
double sum_abs(double *array,int len) {
  double sum;
  int i;

  sum = 0.0;
  for (i=0;i<len;i++) {
    sum += fabs(array[i]);
  }
  return sum;
}
