
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" void start_cuda_profiling() {
  hipProfilerStart();
}

extern "C" void end_cuda_profiling() {
  hipProfilerStop();
}
