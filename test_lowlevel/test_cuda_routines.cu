#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"

class d2 {
 public:
  double d[2];
  
  d2() {this->d[0]=0.0;this->d[1]=0.0;}
  d2(int x) {this->d[0]=x;this->d[1]=0;}
  void setvalue(double x, double y) {d[0]=x; d[1]=y;}
  d2& operator+(d2& x) {
    d2 *res=new d2;
    res->d[0] = this->d[0]+x.d[0];
    res->d[1] = this->d[1]+x.d[1];
    return *res;
  }
  d2& operator+(volatile d2& x) {
    d2 *res=new d2;
    res->d[0] = this->d[0]+x.d[0];
    res->d[1] = this->d[1]+x.d[1];
    return *res;
  }
  d2& operator+=(d2& x) {
    this->d[0] += x.d[0];
    this->d[1] += x.d[1];
    return *this;
  }

  d2& operator=(d2 x) {
    this->d[0]=x.d[0];
    this->d[1]=x.d[1];
    return *this;
  }

  volatile d2& operator=(d2 x) volatile {
    this->d[0]=x.d[0];
    this->d[1]=x.d[1];
    return *this;
  }

  /*  d2& operator=(volatile d2& x) {
    this->d[0]=x.d[0];
    this->d[1]=x.d[1];
    return *this;
    }*/
};

#include "cuda_kernels.cu"
#include "reduction_kernel.cu"

void init_cuda() {
  hipError_t cuda_err;
  int number_of_devices;

  cuda_err = hipGetDeviceCount(&number_of_devices);
  if (cuda_err != hipSuccess) {
    printf("No GPU devices found.\n");
    exit(1);
  }
  if (number_of_devices>=3) {
    cuda_err = hipSetDevice(2);
    if (cuda_err != hipSuccess) {
      printf(hipGetErrorString(cuda_err));
    }
  }
}

void test_cuda_transpose() {
  /* matrix on the host */
  hipfftDoubleComplex *mat;
  hipfftDoubleComplex *tmat;
  dim3 grid;
  dim3 threadblock;
  int lind,klmn,blocksize;
  int width, height,i,j,twidth,theight;
  hipfftDoubleComplex *dev_mat, *dev_tmat;

  width=32;
  height=9;
  blocksize = 1;

  mat = (hipfftDoubleComplex*)malloc(width*height*blocksize*sizeof(hipfftDoubleComplex));
  tmat = (hipfftDoubleComplex*)malloc(width*height*blocksize*sizeof(hipfftDoubleComplex));

  for (klmn=0;klmn<blocksize;klmn++) {
    printf("-------------------- Input block %u ------------------\n",klmn);
    for (i=0;i<height;i++) {
      for (j=0;j<width;j++) {
	lind = klmn*height*width+i*width+j;
	mat[lind].x = (double)i+klmn*100.0;
	mat[lind].y = (double)j+klmn*100.0;
	printf("(%3.0f,%3.0f) ",mat[lind].x,mat[lind].y);
      }
      printf("\n");
    }
  }

  /* allocate on GPU */
  hipMalloc((void**)&dev_mat,width*height*blocksize*sizeof(hipfftDoubleComplex));
  hipMalloc((void**)&dev_tmat,width*height*blocksize*sizeof(hipfftDoubleComplex));

  /* transfer mat to the GPU */
  hipMemcpy(dev_mat,mat,width*height*blocksize*sizeof(hipfftDoubleComplex),hipMemcpyHostToDevice);

  /* call the kernel */
  printf("width = %u, height= %u, TILE_DIM = %u,\nheight/TILE_DIM = %u R %u\n",
	 width, height,TILE_DIM, height/TILE_DIM, height%TILE_DIM);
  grid.x=width/TILE_DIM;

  grid.y = (height%TILE_DIM==0) ? height/TILE_DIM : height/TILE_DIM + 1;
  grid.z = blocksize;
  threadblock.x=TILE_DIM;
  threadblock.y=TILE_DIM;
  printf("grid = (%u,%u,%u), threadblock = (%u,%u,%u)\n",grid.x,grid.y,grid.z,
	 threadblock.x,threadblock.y, threadblock.z);
  transposeCoalescedBank<<<grid,threadblock>>>(dev_tmat,dev_mat,width,height);
    
  for (klmn=0;klmn<blocksize;klmn++) {
    //hipDeviceSynchronize();
    hipMemcpy(&tmat[klmn*width*height],&dev_tmat[klmn*width*height],
	       width*height*sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost);
  }
  twidth = height;
  theight = width;

  for (klmn=0;klmn<blocksize;klmn++) {
    printf("------------------- Output block %u ------------------\n",klmn);
    for (i=0;i<theight;i++) {
      for (j=0;j<twidth;j++) {
	lind = klmn*theight*twidth+i*twidth+j;
	printf("(%3.0f,%3.0f) ",tmat[lind].x,tmat[lind].y);
      }
      printf("\n");
    }
  }
  
  hipFree(dev_mat);
  hipFree(dev_tmat);
  free(mat);
  free(tmat);
}

void test_cuda_maxval_per_block() {
  /* first test the sum reduction */
  double *d_idata, *d_odata;
  double *idata, *odata;
  int i,j,li0,lj0,lbg0,klmn;
  dim3 grid,threadblock;
  int shared_mem, iTile, lines_per_tile,passed;
  int iConfi, iConfj, iConfg;
  const int nConfi = 1;
  const int nConfj = 1;
  const int nConfg = 2;
  //const int Confi[nConfi]={2,7,10,16,24,32,128,384,517};
  const int Confi[nConfi]={5};
  const int Confj[nConfj]={8};
  const int Confg[nConfg]={1,2};
  /*li0 = 24;
  lj0 = 8;
  lbg0 = 2;*/

  /* loop over different configurations */
  for (iConfi=0;iConfi<nConfi;iConfi++) {
    for (iConfj=0;iConfj<nConfj;iConfj++) {
      for (iConfg=0;iConfg<nConfg;iConfg++) {
	li0  = Confi[iConfi];
	lj0  = Confj[iConfj];
	lbg0 = Confg[iConfg];

	// Input arrays on host and device
	idata = (double*)malloc(li0*lj0*lbg0*2*sizeof(double));
	hipMalloc((void**)&d_idata,li0*lj0*lbg0*2*sizeof(double));
	for (klmn=0;klmn<lbg0;klmn++) {
	  for (i=0;i<li0;i++) {
	    for (j=0;j<lj0;j++) {
	      idata[klmn*2*li0*lj0+i*lj0+j] = 1000*klmn+100*j+(double)(i+1);
	      idata[(2*klmn+1)*li0*lj0+i*lj0+j] = 100*klmn+10*j+80.0-(double)(0.1*(i+1));
	      //printf("(%7.1f, %7.1f)",idata[klmn*2*li0*lj0+i*lj0+j],idata[(2*klmn+1)*li0*lj0+i*lj0+j]);
	    }
	    //printf("\n");
	  }
	  //printf("-----------\n");
	}
	hipMemcpy(d_idata,idata,li0*lj0*lbg0*2*sizeof(double),hipMemcpyHostToDevice);
	
	// Output arrays on host and device
	odata = (double*)malloc(lbg0*2*NTILES*sizeof(double));
	hipMalloc((void**)&d_odata,lbg0*2*NTILES*sizeof(double));
	
	grid.x=lbg0;
	grid.y=NTILES;
	threadblock.x=lj0;
	threadblock.y=1;
	shared_mem = (li0<64 ? 64 : li0)*2*sizeof(double);
	cuda_maxval_per_block<<<grid,threadblock,shared_mem>>>(d_idata, d_odata,li0, lj0);
	
	// Transfer the result from device to host
	hipMemcpy(odata,d_odata,2*lbg0*NTILES*sizeof(double),hipMemcpyDeviceToHost);
	
	passed = 1;
	lines_per_tile = li0/NTILES;
	for (klmn=0;klmn<lbg0;klmn++) {
	  for (iTile=0;iTile<NTILES;iTile++) {
#if 1
	    if ((odata[klmn*NTILES+iTile] != 1000*klmn+100*(lj0-1)+(double)(iTile+1)*lines_per_tile) ||
		(odata[lbg0*NTILES+klmn*NTILES+iTile] != 100*klmn+10*(lj0-1)+80.0-0.1*(iTile*lines_per_tile+1))) {
	      passed=0;
	      printf("klmn=%u, tile %u: %f\t%f, reference is %f\t%f\n",
		     klmn,iTile,odata[klmn*NTILES+iTile],odata[lbg0*NTILES+klmn*NTILES+iTile],
		     1000*klmn+100*(lj0-1)+(double)(iTile+1)*lines_per_tile,
		     100*klmn+10*(lj0-1)+80.0-0.1*(iTile*lines_per_tile+1));
	    }
#else
	    printf("klmn=%u, tile %u: %f\t%f, reference is %f\t%f\n",
		   klmn,iTile,odata[klmn*NTILES+iTile],odata[lbg0*NTILES+klmn*NTILES+iTile],
		   1000*klmn+100*(lj0-1)+(double)(iTile+1)*lines_per_tile,
		   100*klmn+10*(lj0-1)+80.0-0.1*(iTile*lines_per_tile+1));
#endif
	  }
	}
	printf("li0 = %u, lj0 = %u, lbg0 = %u: ",li0,lj0,lbg0);
	if (passed) printf("passed.\n");
	else printf("\tFAILED!\n");
	
	hipFree(d_odata);
	hipFree(d_idata);
	free(odata);
	free(idata);
      }
    }
  }
}

void test_cuda_reduction() {

  /* first test the sum reduction */
  double *d_idata, *d_odata, *d_redres0, *d_redres1;
  double *idata, *odata,redres0,redres1;
  int i,li0,lj0,lbg0,j,klmn;
  int whichKernel = 6;
  dim3 grid,threadblock;
  int shared_mem;
  li0 = 32;
  lj0 = 8;
  lbg0 = 16;

  /*size = 4096;
  threads = 128;
  blocks = size/threads;
  printf("We are using %u blocks with %u threads,\n",blocks,threads);*/

  // Input arrays on host and device
  idata = (double*)malloc(li0*lj0*lbg0*2*sizeof(double));
  hipMalloc((void**)&d_idata,li0*lj0*lbg0*2*sizeof(double));
  for (klmn=0;klmn<lbg0;klmn++) {
    for (i=0;i<li0;i++) {
      for (j=0;j<lj0;j++) {
	idata[klmn*2*li0*lj0+i*lj0+j] = 1000*klmn+100*j+(double)(i+1);
	idata[(2*klmn+1)*li0*lj0+i*lj0+j] = 100*klmn+10*j+80.0-(double)(0.1*(i+1));
	//printf("(%f, %f)",idata[klmn*2*li0*lj0+i*lj0+j],idata[(2*klmn+1)*li0*lj0+i*lj0+j]);
      }
      //printf("\n");
    }
    //printf("-----------\n");
  }
  hipMemcpy(d_idata,idata,li0*lj0*lbg0*2*sizeof(double),hipMemcpyHostToDevice);

  // Output arrays on host and device
  odata = (double*)malloc(lbg0*2*sizeof(double));
  hipMalloc((void**)&d_odata,lbg0*2*sizeof(double));
  hipMalloc((void**)&d_redres0,sizeof(double));
  hipMalloc((void**)&d_redres1,sizeof(double));

  grid.x=lbg0;
  threadblock.x=li0;
  threadblock.y=1;
  shared_mem = (li0<64 ? 64 : li0)*2*sizeof(double);
  cuda_maxval_per_block<<<grid,threadblock,shared_mem>>>(d_idata, d_odata,li0, lj0);

  //reduce<double>(size,threads,blocks,whichKernel, d_idata, d_odata,REDUCTION_MAX);

  // Next level reduce
  /*if (whichKernel==6) {
    size=blocks/2;
  } else {
    size = blocks;
    }*/
  reduce<double>(lbg0,whichKernel,d_odata,d_redres0,REDUCTION_MAX,0);
  reduce<double>(lbg0,whichKernel,d_odata,d_redres1,REDUCTION_MAX,lbg0);

  // Transfer the result from device to host
  hipMemcpy(odata,d_odata,2*lbg0*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(&redres0,d_redres0,sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(&redres1,d_redres1,sizeof(double),hipMemcpyDeviceToHost);

  // print the result of the reduction
  // for comparison sum i_{i=1}^N=0.5*(N+1)*N
  printf("Intermediate result:\n");
  for (klmn=0;klmn<lbg0;klmn++) {
    printf("%f %f\n",odata[klmn],odata[lbg0+klmn]);
  }
  printf("Final result: %f %f\n",redres0, redres1);
  
}

int main(int argc, char *argv[]) {
  /* test the transpose on the GPU */

  init_cuda();
  test_cuda_transpose();
  //test_cuda_maxval_per_block();
  //test_cuda_reduction();
}

