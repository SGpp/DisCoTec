#include "hip/hip_runtime.h"
#include "cuda_overlap.h"
#include "cuda_kernels.h"

/* The following routine has been found in the web at
   http://brianmykietka.net/projects.php?project=finalmatrixtranspose
   It references a paper "Optimizing Matrix Transpose in CUDA" by
   Greg Ruetsch (gruetsch@nvidia.com) and Paulius Micikevicius (pauliusm@nvidia.com)
*/


/* odata, output array, which is transposed
   idata, input array of dimension width x height
   width
   height
*/
#if 0
__global__ void transposeCoalescedBank_old(hipfftDoubleComplex* odata, 
					   hipfftDoubleComplex* idata, 
					   int width, int height)
{
	__shared__ hipfftDoubleComplex tile[TILE_DIM][TILE_DIM + 1];
	
	int klmn = blockIdx.z;
	int offset = klmn*width*height;
	int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
	int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
	int index_in = offset + yIndex * width + xIndex ;
	
	/* The same for the output matrix, which is transposed. */
	xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
	yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
	int index_out = offset + yIndex * height + xIndex ;
	
	/* copy one tile per threadblock from global memory to shared memory */
	for (int i = 0; i < TILE_DIM; i += blockDim.y) {
	  tile[ threadIdx.y + i ][ threadIdx.x ] =
	    idata[ index_in + i * width];
	}
	
	__syncthreads();
	
	for (int i = 0; i < TILE_DIM; i += blockDim.y) {
	  odata[ index_out + i * height ] =
	    tile[ threadIdx.x ][ threadIdx.y + i ];
	}
}
#endif
__global__ void transposeCoalescedBank(hipfftDoubleComplex* odata, 
				       hipfftDoubleComplex* idata, 
				       int width, int height, int tile_dim)
{
  /*__shared__ hipfftDoubleComplex tile[TILE_DIM][TILE_DIM + 1];*/
  extern __shared__ hipfftDoubleComplex tile[];
	
  int klmn = blockIdx.z;
  int offset = klmn*width*height;
  int xIndex_in = blockIdx.x * tile_dim + threadIdx.x;
  int yIndex_in = blockIdx.y * tile_dim + threadIdx.y;
  int index_in = offset + yIndex_in * width + xIndex_in ;
	
  /* The same for the output matrix, which is transposed. */
  int xIndex = blockIdx.y * tile_dim + threadIdx.x;
  int yIndex = blockIdx.x * tile_dim + threadIdx.y;
  int index_out = offset + yIndex * height + xIndex ;
	
  /* copy one tile per threadblock from global memory to shared memory */
  if (yIndex_in < height) {
    /*printf("(%u,%u), (%u,%u), %2u %2u %2u | %2u %2u %2u\n",
      blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,
      xIndex_in, yIndex_in, index_in, 
      xIndex, yIndex, index_out);*/
    /*tile[ threadIdx.y ][ threadIdx.x ] =
      idata[ index_in ];*/
    tile[ threadIdx.y*(tile_dim+1) + threadIdx.x ] =
      idata[ index_in ];
  }
	
  __syncthreads();
	
  if (xIndex<height) {
    /*odata[ index_out ] =
      tile[ threadIdx.x ][ threadIdx.y ];*/
    odata[ index_out ] =
      tile[ threadIdx.x *(tile_dim+1) + threadIdx.y ];
  }
}

void transpose_wrapper(hipfftDoubleComplex* odata, 
		       hipfftDoubleComplex* idata, 
		       int width, int height, int nXYPlanesPerPart, int iStream) {
  dim3 grid, threadblock;
  hipError_t cuda_err;
  int minimum, max_tile_dim,eval_tile_dim,tile_dim;

  /* determine dynamically the TILE_DIM,
     it should be in the range 2->31. At the moment, we assume that
     width is a power of 2. */
  minimum=(width<height) ? width : height;
  max_tile_dim=16;
  if (minimum<16) max_tile_dim=8;
  if (minimum<8) max_tile_dim=4;
  if (minimum<4) max_tile_dim=2;

  for (eval_tile_dim=max_tile_dim;eval_tile_dim>1;eval_tile_dim=eval_tile_dim>>1) {
    if (width%eval_tile_dim==0) {
      tile_dim=eval_tile_dim;
      break;
    }
  }

  if (height%tile_dim) grid.y=height/tile_dim + 1;
  else grid.y=height/tile_dim;

  grid.x=width/tile_dim;   grid.z = nXYPlanesPerPart;
  threadblock.x=tile_dim;  threadblock.y=tile_dim;  threadblock.z=1;
  
  /*printf("Starting transposeCoalescedBank<<<(%u,%u,%u),(%u,%u,%u)>>>(,,%u,%u).\n",
	 grid.x,grid.y,grid.z,
	 threadblock.x,threadblock.y,threadblock.z,
	 width,height);*/
  transposeCoalescedBank<<<grid,threadblock,tile_dim*(tile_dim+1)*sizeof(hipfftDoubleComplex),mystreams[iStream]->cudaStream>>>(odata,idata, width,height,tile_dim);
  
  cuda_err = hipGetLastError();
  if (cuda_err != hipSuccess) {
    printf("Error with kernel launch.(transposeCoalescedBank)\n");
    printf("%s\n",hipGetErrorString(cuda_err));
  }
}

/* gridDim.x = nXYPlanesPerPart;
   gridDim.y = nj0;
   blockDim.x = li0da;
*/
__global__ void copy_with_pnl(hipfftDoubleComplex *inarr, hipfftDoubleComplex *outarr, hipfftDoubleReal *prefactor, int width, int height) {
  /* inarr[nXYPlanesPerPart][nj0][li0da];
     outarr[nXYPlanesPerPart][nj0][li0da];
     prefactor[li0da];
  */
  int index = blockIdx.x*width*height + blockIdx.y*width + threadIdx.x;

  outarr[index].x += inarr[index].x * prefactor[threadIdx.x];
  outarr[index].y += inarr[index].y * prefactor[threadIdx.x];

}


/* Copies the inarr to a larger array (larger in height nj0 ->ly0da/2+1).
   Rows 0->nj0 are copied, nj0+1->ly0da/2+1 are set to zero.
   One threadblock per x-y plane and one thread for each point in the x-y plane
   of the larger output array.
   PROBLEM: This easily exceeds 1024 threads per block.
   Therefore, one block just works on one row.*/
__global__ void dev_copy_and_zero_for_dealiasing_new(hipfftDoubleComplex *inarr, int iwidth, int iheight,
						     hipfftDoubleComplex *outarr,int owidth, int oheight) {
  /* double _Complex inarr[howmany][iheight][iwidth];
     double _Complex outarr[howmany][oheight][owidth];
     gridDim.x = oheight; gridDim.y = howmany;
     blockDim.x = owidth; blockDim.y=1; 
  */
  int iOffset = blockIdx.z*iwidth*iheight+blockIdx.x*iwidth;
  int oOffset = blockIdx.z*owidth*oheight+blockIdx.x*owidth;

  int ilind;
  int olind = oOffset + threadIdx.x;

  if (blockIdx.x < iheight) {
    /* just copy */
    ilind = iOffset+threadIdx.x;
    outarr[olind] = inarr[ilind];
  } else {
    /* set entry to zero, x is real part, y is imaginary part. */
    outarr[olind].x = 0.0;
    outarr[olind].y = 0.0;
  }
}


#ifdef WITH_THRUST

/* does not compile due to some errors in the include files.
   but it is not suitable for GENE, as it does not support different
   CUDA streams, which is crucial for GENE performance. */
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/functional.h>
//#include <thrust/device_vector.h>
/*#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
*/

/* first, we declare a transformation for the indices by deriving it from
   the unary_function */
struct skip_zero_transform_t : public thrust::unary_function<int,int> {

  int dimXYPlane;
  int dimExtendedXYPlane;

  /* the constructor initializes the two attributes of the struct */
  skip_zero_transform_t(int _dimXYPlane,int _dimExtendedXYPlane) : dimXYPlane(_dimXYPlane),dimExtendedXYPlane(_dimExtendedXYPlane) {}

  __host__ __device__
  int operator()(const int i)  const {
    return (i/dimXYPlane)*dimExtendedXYPlane+(i%dimXYPlane);
  }
};

/* and the complementary function */
struct only_zero_transform_t : public thrust::unary_function<int,int> {

  int dimXYPlane;
  int dimExtendedXYPlane;
  int dimExtension;

  /* the constructor initializes the two attributes of the struct */
  only_zero_transform_t(int _dimXYPlane,int _dimExtendedXYPlane) : dimXYPlane(_dimXYPlane),
								   dimExtendedXYPlane(_dimExtendedXYPlane) {
    dimExtension = _dimExtendedXYPlane-_dimXYPlane;
  }

  __host__ __device__
  int operator()(const int i) const {
    return dimXYPlane + (i/dimExtension)*dimExtendedXYPlane + (i%dimExtension);
  }
};


void copy_and_zero_for_dealiasing_wrapper(hipfftDoubleComplex *inarr, int iwidth, int iheight,int idepth,
					  hipfftDoubleComplex *outarr, int owidth, int oheight, int iStream) {

  hipfftDoubleComplex zero_value;
  zero_value.x=0.0;
  zero_value.y=0.0;
  thrust::device_ptr<hipfftDoubleComplex> dptr_inarr=thrust::device_pointer_cast(inarr);
  thrust::device_ptr<hipfftDoubleComplex> dptr_outarr=thrust::device_pointer_cast(outarr);

  thrust::device_vector<hipfftDoubleComplex> dv_inarr(dptr_inarr,dptr_inarr+iwidth*iheight*idepth);
  thrust::device_vector<hipfftDoubleComplex> dv_outarr(dptr_outarr,dptr_outarr+owidth*oheight*idepth);

  thrust::counting_iterator<int> counter(0);
  skip_zero_transform_t skip_zero(iwidth*iheight,owidth*oheight); // instantiate
  only_zero_transform_t only_zero(iwidth*iheight,owidth*oheight);

  /* copy the inarr to the outarr, but only for the first iwidth*iheight entries per
     xy-plane. This scattering is done with the permutation_iterator. */
  //thrust::copy(dptr_inarr,dptr_inarr+iwidth*iheight*idepth,
  thrust::copy(dv_inarr.begin(),dv_inarr.end(),
	       thrust::make_permutation_iterator(dv_outarr.begin(),
						 thrust::make_transform_iterator(counter, skip_zero)
						 )
	       );

  thrust::fill_n(thrust::make_permutation_iterator(dv_outarr.begin(),
						 thrust::make_transform_iterator(counter, only_zero)
						 ),
	       (owidth*oheight-iwidth*iheight)*idepth,
	       zero_value);
}
#else
/* non-thrust version */
void copy_and_zero_for_dealiasing_wrapper(hipfftDoubleComplex *inarr, int iwidth, int iheight,int idepth,
					  hipfftDoubleComplex *outarr, int owidth, int oheight, int iStream) {

  dim3 grid, threadblock;

  //if (iwidth<maxThreadsPerBlock) {
    /* at least one line matches in the threadblock */
    //nLinesPerTile=maxThreadsPerBlock/iwidth;

  //grid.x=ly0da/2+1;        grid.y=(ly0da/2+1)/4;  grid.z=idepth;
  grid.x=ly0da/2+1;        grid.y=1;  grid.z=idepth;
  //threadblock.x=iwidth;    threadblock.y=4;
  threadblock.x=iwidth;    threadblock.y=1;
    
  //printf("before dev_copy_and_zero_for_dealiasing_new: %s\n",hipGetErrorString(hipGetLastError()));
  dev_copy_and_zero_for_dealiasing_new<<<grid,threadblock,0,mystreams[iStream]->cudaStream>>>
    (inarr,iwidth,iheight,  outarr,iwidth,ly0da/2+1);
  //hipStreamSynchronize(mystreams[iStream]->cudaStream);
  //printf("after dev_copy_and_zero_for_dealiasing_new: %s\n",hipGetErrorString(hipGetLastError()));

}
#endif


/* find the maximal value in arr and return it in maxval,
   this function is specialized for the use with GENE, so it
   assumes the input array arr to have the dimensions:
   hipfftDoubleReal arr[lbg0][2][li0da][ly0da];
   It finds the maximum individually for the two dimensions of the
   second index.

   gridDim.x=lbg0;
   blockDim.x = li0da;
   blockDim.y = 1;

   This routines needs a shared memory of 
   shared_mem = (li0<64 ? 64 : li0da)*2*sizeof(double);

*/
#if 0
__global__ void cuda_maxval_per_block_old(const hipfftDoubleReal *arr, hipfftDoubleReal *g_odata,
				      const int li0da, const int ly0da) {
  /* declaring extern the shared memory array means that it is allocated at launch time. */

  extern __shared__ hipfftDoubleReal sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    /*unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
      unsigned int gridSize = blockSize*2*gridDim.x;*/
    unsigned int blockSize = blockDim.x*blockDim.y*blockDim.z;
    int j, lind;
    hipfftDoubleReal *sdata0 = &sdata[0];
    hipfftDoubleReal *sdata1 = &sdata[blockDim.x];

    hipfftDoubleReal myMax[2] = {0.0,0.0};
    
    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    for (j=0;j<ly0da;j++) {
      lind = blockIdx.x*2*ly0da*li0da+threadIdx.x*ly0da+j;
      if (arr[lind]>myMax[0]) {
	myMax[0]=arr[lind];
      }

      lind = (blockIdx.x*2+1)*ly0da*li0da+threadIdx.x*ly0da+j;
      if (arr[lind]>myMax[1]) {
	myMax[1]=arr[lind];
      }
    }

    // each thread puts its local sum into shared memory 
    sdata0[tid] = myMax[0];
    sdata1[tid] = myMax[1];
    __syncthreads();

    /* Now we have the maximum of each line in the shared memory. */
    // do reduction in shared mem
    if (blockSize >= 512) { 
      if (tid < 256) { 
	sdata0[tid] = (sdata0[tid]>sdata0[tid+256]) ? sdata0[tid] : sdata0[tid+256]; 
	sdata1[tid] = (sdata1[tid]>sdata1[tid+256]) ? sdata1[tid] : sdata1[tid+256]; 
      } __syncthreads(); 
    }
    if (blockSize >= 256) { 
      if (tid < 128) { 
	sdata0[tid] = (sdata0[tid]>sdata0[tid+128]) ? sdata0[tid] : sdata0[tid+128]; 
	sdata1[tid] = (sdata1[tid]>sdata1[tid+128]) ? sdata1[tid] : sdata1[tid+128]; 
      } __syncthreads(); 
    }
    if (blockSize >= 128) { 
      if (tid <  64) { 
	sdata0[tid] = (sdata0[tid]>sdata0[tid+64]) ? sdata0[tid] : sdata0[tid+64]; 
	sdata1[tid] = (sdata1[tid]>sdata1[tid+64]) ? sdata1[tid] : sdata1[tid+64]; 
      } __syncthreads(); 
    }
	
    
    if (tid < 32) {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
      volatile hipfftDoubleReal* smem = sdata0;
      if (blockSize >=  64) { smem[tid] = (smem[tid]>smem[tid+32]) ? smem[tid] : smem[tid+32]; }
      if (blockSize >=  32) { smem[tid] = (smem[tid]>smem[tid+16]) ? smem[tid] : smem[tid+16]; }
      if (blockSize >=  16) { smem[tid] = (smem[tid]>smem[tid+8])  ? smem[tid] : smem[tid+8]; }
      if (blockSize >=  8)  { smem[tid] = (smem[tid]>smem[tid+4])  ? smem[tid] : smem[tid+4]; }
      if (blockSize >=  4)  { smem[tid] = (smem[tid]>smem[tid+2])  ? smem[tid] : smem[tid+2]; }
      if (blockSize >=  2)  { smem[tid] = (smem[tid]>smem[tid+1])  ? smem[tid] : smem[tid+1]; }
      smem = sdata1;
      if (blockSize >=  64) { smem[tid] = (smem[tid]>smem[tid+32]) ? smem[tid] : smem[tid+32]; }
      if (blockSize >=  32) { smem[tid] = (smem[tid]>smem[tid+16]) ? smem[tid] : smem[tid+16]; }
      if (blockSize >=  16) { smem[tid] = (smem[tid]>smem[tid+8]) ? smem[tid] : smem[tid+8]; }
      if (blockSize >=  8) { smem[tid] = (smem[tid]>smem[tid+4]) ? smem[tid] : smem[tid+4]; }
      if (blockSize >=  4) { smem[tid] = (smem[tid]>smem[tid+2]) ? smem[tid] : smem[tid+2]; }
      if (blockSize >=  2) { smem[tid] = (smem[tid]>smem[tid+1]) ? smem[tid] : smem[tid+1]; }
    }
    
    // write result for this block to global mem 
    if (tid == 0)  {
      g_odata[blockIdx.x] = sdata0[0];
      g_odata[gridDim.x+blockIdx.x] = sdata1[0];
    }
}
#endif

/** Finding the maximum value of an x-y plane for the two parts of an array
    separately.

    Each column of the input array is reduced by one thread and in one tile, that
    means over li0da/NTILES rows.

    \param arr The input array. The structure of this array is [lbg0][2][li0da][ly0da].
    \param g_odata The output array. It contains after the routine the result and is of the
    structure [2][lbg0][NTILES]. It is then suitable for usage of the reduction kernel
    for the two separate parts to find the global maximum.
    \param li0da The number of rows of an x-y plane (the x direction).
    \param ly0da The number of columns of a x-y plane (the y direction).

    The launch parameters should be:
    grid.x = lbg0
    grid.y = NTILES
    grid.z = 1
    threadblock.x = ly0da, each column is one thread
    threadblock.y = 1
    threadblock.z = 1
    shared_mem = ((ly0da<32) ? 64 : 2*ly0da)*sizeof(double);

    \constraint li0da must be divisable by NTILES
 */
__global__ void cuda_maxval_per_block(const hipfftDoubleReal *arr, hipfftDoubleReal *g_odata,
				      const int li0da, const int ly0da) {
  /* declaring extern the shared memory array means that it is allocated at launch time. */

  extern __shared__ hipfftDoubleReal sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    /*unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
      unsigned int gridSize = blockSize*2*gridDim.x;*/
    unsigned int blockSize = max(blockDim.x,32);//blockDim.x;
    int lind, iXYPlane,iBlockrow,compare_lind, reducedToPowerOfTwo;
    int diff;
    hipfftDoubleReal *sdata0 = &sdata[0];
    hipfftDoubleReal *sdata1 = &sdata[blockDim.x];

    //hipfftDoubleReal myMax[2] = {0.0,0.0};
    
    iXYPlane = blockIdx.x*2*li0da*ly0da;
    iBlockrow = blockIdx.y*ly0da*li0da/NTILES;
    //lind = iXYPlane + iBlockrow + threadIdx.y*ly0da + threadIdx.x;
    lind = iXYPlane + iBlockrow + threadIdx.x;

    /* Now compare one row with the li0da/NTILES-1 rows in the tile. */
    compare_lind = lind+ly0da;
    sdata0[tid] = (arr[lind]>arr[compare_lind]) ? arr[lind] : arr[compare_lind];
    compare_lind += ly0da;
    while (compare_lind < lind+ly0da*li0da/NTILES) {
      sdata0[tid] = (sdata0[tid]>arr[compare_lind]) ? sdata0[tid] : arr[compare_lind];
      compare_lind += ly0da;
    }
    /* In sdata0 the column maxima over the tile are stored now. */

    iXYPlane = (2*blockIdx.x+1)*li0da*ly0da;
    iBlockrow = blockIdx.y*ly0da*li0da/NTILES;
    //lind = iXYPlane + iBlockrow + threadIdx.y*ly0da + threadIdx.x;
    lind = iXYPlane + iBlockrow + threadIdx.x;

    compare_lind = lind+ly0da;
    sdata1[tid] = (arr[lind]>arr[compare_lind]) ? arr[lind] : arr[compare_lind];
    compare_lind += ly0da;
    while (compare_lind < lind+ly0da*li0da/NTILES) {
      sdata1[tid] = (sdata1[tid]>arr[compare_lind]) ? sdata1[tid] : arr[compare_lind];
      compare_lind += ly0da;
    }


    // each thread puts its local sum into shared memory 
    __syncthreads();

    /* Now we have one line with the maxima of each column in the shared memory. 
       BUT, the number of threads and the blockSize can be arbitrary (usually
       divisable by 2, but not necessarily a power of 2), so we have to modify the
       algorithm. */
    
    // do reduction in shared mem
    reducedToPowerOfTwo=0;
    if (blockSize >= 512) { 
      diff = blockSize-512;
      if (tid < diff) {
	sdata0[tid] = (sdata0[tid]>sdata0[512+tid]) ? sdata0[tid] : sdata0[512+tid]; 
	sdata1[tid] = (sdata1[tid]>sdata1[tid+512]) ? sdata1[tid] : sdata1[tid+512]; 
      }
      __syncthreads();
      reducedToPowerOfTwo=1;
      if (tid < 256) { 
	sdata0[tid] = (sdata0[tid]>sdata0[tid+256]) ? sdata0[tid] : sdata0[tid+256]; 
	sdata1[tid] = (sdata1[tid]>sdata1[tid+256]) ? sdata1[tid] : sdata1[tid+256]; 
      } __syncthreads(); 
    }

    if (blockSize >= 256) { 
      diff = blockSize-256;
      if (!reducedToPowerOfTwo && (tid < diff)) {
	sdata0[tid] = (sdata0[tid]>sdata0[tid+256]) ? sdata0[tid] : sdata0[tid+256]; 
	sdata1[tid] = (sdata1[tid]>sdata1[tid+256]) ? sdata1[tid] : sdata1[tid+256]; 
      } __syncthreads();
      reducedToPowerOfTwo=1;

      if (tid < 128) { 
	sdata0[tid] = (sdata0[tid]>sdata0[tid+128]) ? sdata0[tid] : sdata0[tid+128]; 
	sdata1[tid] = (sdata1[tid]>sdata1[tid+128]) ? sdata1[tid] : sdata1[tid+128]; 
      } __syncthreads(); 
    }
    if (blockSize >= 128) { 
      diff = blockSize - 128;
      if (!reducedToPowerOfTwo && (tid < diff)) {
	sdata0[tid] = (sdata0[tid]>sdata0[tid+128]) ? sdata0[tid] : sdata0[tid+128]; 
	sdata1[tid] = (sdata1[tid]>sdata1[tid+128]) ? sdata1[tid] : sdata1[tid+128]; 
      } __syncthreads();
      reducedToPowerOfTwo=1;
      if (tid <  64) { 
	sdata0[tid] = (sdata0[tid]>sdata0[tid+64]) ? sdata0[tid] : sdata0[tid+64]; 
	sdata1[tid] = (sdata1[tid]>sdata1[tid+64]) ? sdata1[tid] : sdata1[tid+64]; 
      } __syncthreads(); 
    }
	
    if (!reducedToPowerOfTwo && (blockSize>32)) {
      diff = blockSize-32;
      if (tid<diff) {
	sdata0[tid] = (sdata0[tid]>sdata0[tid+32]) ? sdata0[tid] : sdata0[tid+32]; 
	sdata1[tid] = (sdata1[tid]>sdata1[tid+32]) ? sdata1[tid] : sdata1[tid+32]; 
      }
    }
    if (tid < 32) {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
      volatile hipfftDoubleReal* smem = sdata0;
      if (blockSize >=  64) { smem[tid] = (smem[tid]>smem[tid+32]) ? smem[tid] : smem[tid+32]; }
      if (blockSize >=  32) { smem[tid] = (smem[tid]>smem[tid+16]) ? smem[tid] : smem[tid+16]; }
      if (blockSize >=  16) { smem[tid] = (smem[tid]>smem[tid+8])  ? smem[tid] : smem[tid+8]; }
      if (blockSize >=  8)  { smem[tid] = (smem[tid]>smem[tid+4])  ? smem[tid] : smem[tid+4]; }
      if (blockSize >=  4)  { smem[tid] = (smem[tid]>smem[tid+2])  ? smem[tid] : smem[tid+2]; }
      if (blockSize >=  2)  { smem[tid] = (smem[tid]>smem[tid+1])  ? smem[tid] : smem[tid+1]; }
      smem = sdata1;
      if (blockSize >=  64) { smem[tid] = (smem[tid]>smem[tid+32]) ? smem[tid] : smem[tid+32]; }
      if (blockSize >=  32) { smem[tid] = (smem[tid]>smem[tid+16]) ? smem[tid] : smem[tid+16]; }
      if (blockSize >=  16) { smem[tid] = (smem[tid]>smem[tid+8]) ? smem[tid] : smem[tid+8]; }
      if (blockSize >=  8) { smem[tid] = (smem[tid]>smem[tid+4]) ? smem[tid] : smem[tid+4]; }
      if (blockSize >=  4) { smem[tid] = (smem[tid]>smem[tid+2]) ? smem[tid] : smem[tid+2]; }
      if (blockSize >=  2) { smem[tid] = (smem[tid]>smem[tid+1]) ? smem[tid] : smem[tid+1]; }
    }
    
    // write result for this block to global mem 
    if (tid == 0)  {
      g_odata[blockIdx.x*gridDim.y+blockIdx.y] = sdata0[0];
      g_odata[ gridDim.x*gridDim.y+blockIdx.x*gridDim.y+blockIdx.y] = sdata1[0];
    }
}

/* compute the standard nonlinearity
   1. Attempt:
   grid.x = lbg0
   grid.y = NTILES_NONLIM
   threadblock.x=ly0da
   threadblock.y=li0da/NTILES_NONLIM
*/
__global__ void comp_stand_nonlin(double *vexy_re, double *dgdxy_re, double* nonlin1_re,int li0da, int ly0da) {
  /* offset0 is the start index of the first component for the given threadblock */
  int offset0 =  2*blockIdx.x   *li0da*ly0da + blockIdx.y*blockDim.y*blockDim.x;
  int offset1 = (2*blockIdx.x+1)*li0da*ly0da + blockIdx.y*blockDim.y*blockDim.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  //for (i=0;i<li0da;i++) {
  nonlin1_re[ blockIdx.x*li0da*ly0da + blockIdx.y*blockDim.y*blockDim.x + ty*ly0da + tx ] = 
    - vexy_re[ offset0 + ty*ly0da + tx ]*dgdxy_re[ offset1+ty*ly0da + tx ]
    + vexy_re[ offset1 + ty*ly0da + tx ]*dgdxy_re[ offset0+ty*ly0da + tx ];
    //}
}

/* compute the standard nonlinearity
   1. Attempt:
   grid.x = lbg0
   threadblock.x=ly0da
*/
#if 0
__global__ void comp_stand_nonlin_old(double *vexy_re, double *dgdxy_re, double* nonlin1_re,int li0da, int ly0da) {
  /* offset0 is the start index of the first component for the given threadblock */
  int offset0 =  2*blockIdx.x   *li0da*ly0da;
  int offset1 = (2*blockIdx.x+1)*li0da*ly0da;

  int i;

  /*for (j=0;j<ly0da;j++) {
    nonlin1_re[ blockIdx.x*li0da*ly0da + threadIdx.x*ly0da + j ] = 
      - vexy_re[offset0+threadIdx.x*ly0da+j]*dgdxy_re[offset1+threadIdx.x*ly0da+j]
      + vexy_re[offset1+threadIdx.x*ly0da+j]*dgdxy_re[offset0+threadIdx.x*ly0da+j];
      }*/
  for (i=0;i<li0da;i++) {
    nonlin1_re[ blockIdx.x*li0da*ly0da + i*ly0da + threadIdx.x ] = 
      - vexy_re[ offset0 + i*ly0da + threadIdx.x ]*dgdxy_re[ offset1+i*ly0da + threadIdx.x ]
      + vexy_re[ offset1 + i*ly0da + threadIdx.x ]*dgdxy_re[ offset0+i*ly0da + threadIdx.x ];
  }
}
#endif

void comp_stand_nonlin_wrapper(double *vexy_re, double *dgdxy_re, double *nonlin1_re, 
			       int li0da, int ly0da, int iStream) {
  dim3 grid, threadblock;
  hipError_t cuda_err;
  int nXYPoints,nTiles,nLinesPerTile;

  nXYPoints = li0da*ly0da;
  if (nXYPoints<=maxThreadsPerBlock) {
    grid.x=lbg0;
    grid.y=1;
    threadblock.x=ly0da;
    threadblock.y=li0da;
  } else {
    /* separate the height in several tiles */
    nLinesPerTile = maxThreadsPerBlock/ly0da;
    while (li0da%nLinesPerTile) nLinesPerTile--;
    nTiles = li0da/nLinesPerTile;
    
    if (li0da%nTiles) {
      printf("li0da (%u) must be divisable by nTiles (%u)\n",li0da,nTiles);
    }
    //printf("nLinesPerTile = %u, nTiles = %u\n",nLinesPerTile,nTiles);
    grid.x=lbg0;
    grid.y=nTiles;
    threadblock.x=ly0da;
    threadblock.y=li0da/nTiles;
  }

  /*grid.x=lbg0;
  grid.y=NTILES_NONLIN;
  threadblock.x=ly0da;
  threadblock.y=li0da/NTILES_NONLIN;*/
  /*printf("Calling kernel comp_stand_nonlin with grid(%u,%u,%u) and block(%u,%u,%u), li0da=%u, ly0da=%u.\n",
    grid.x,grid.y,grid.z,threadblock.x,threadblock.y,threadblock.z,li0da,ly0da);*/
  comp_stand_nonlin<<<grid,threadblock,0,mystreams[iStream]->cudaStream>>>(vexy_re,dgdxy_re,nonlin1_re,li0da,ly0da);
  cuda_err = hipGetLastError();
  if (cuda_err != hipSuccess) {
    printf("Error with kernel launch.(comp_stand_nonlin)\n");
    printf("%s\n",hipGetErrorString(cuda_err));
  }
}
