#include "hip/hip_runtime.h"
#include <complex.h>
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "cuda_overlap.h"

extern int nj0;
extern int lbg0;
extern double sum_cabs(hipfftDoubleComplex *array,int len);
extern double sum_cabs(double _Complex *array,int len);
//extern hipStream_t stream[nStreams];
extern Streamdata *mystreams[nStreams];
extern unsigned long int allocatedDeviceMemory;

/*
Module fourier
  USE discretization, ONLY: ni0,nj0,li0,lj0,li1,li2,lj1,lj2,xy_local,y_local,n_procs_x,n_procs_y,mype,my_pex,yx_order
  USE communications, ONLY: mpi_comm_x,MPI_COMPLEX_TYPE
  USE par_in, ONLY: fourier2D
  Implicit None

  Include "fftw3.f"

  public ::  initialize_fourier,fft_kx_to_x,fft_ky_to_y,fft_ff_to_xy,fft_ff_to_xy_t,&
       fft_y_to_ky,fft_x_to_kx,fft_xy_to_ff,fft_xy_to_ff_t,&
       to_real_y,to_fourier_y,finalize_fourier,&
       initialize_fourier_x_1d,to_fourier_x_1d,to_real_x_1d,finalize_fourier_x_1d,&
       check_fourier, initialize_fourier_boundary, to_real_boundary,to_fourier_boundary,&
       finalize_fourier_boundary

  private
*/
/*Integer(Kind=8):: fplan_x, bplan_x, fplan_y, bplan_y, fplan_xy, bplan_xy
  Integer(Kind=8):: bplan_x_1d, fplan_x_1d
  Real:: facnnx, facnny, facbound
  Integer:: inc_1x, num_x, ub_out_x, li0da, ly0da*/

static int li0da, ly0da;
static hipfftHandle *r2c_plan_y_block, *c2r_plan_y_block;
//static hipfftDoubleComplex *dev_cmplxarr;
//static hipfftDoubleComplex *dev_cmplxblock_with_da;
static hipfftDoubleComplex **dp_temp_with_da;
static double facnny;

extern "C" long int cuda_fourier_get_memory_need_on_device(int a_li0da, int a_ly0da) {
  long int mem_need;

  mem_need = 0L;
  /* dev_cmplxblock_with_da */
  //mem_need = lbg0*li0da*(ly0da/2+1)*sizeof(hipfftDoubleComplex);

  /* dp_temp_with_da[iStream] */
  mem_need += nStreams*2*lbg0*a_li0da*(a_ly0da/2+1)*sizeof(hipfftDoubleComplex)/nParts;

  return mem_need;
}

/*!
  !	Initialization for fourier routines
  !*/
extern "C" void initialize_fourier_cufft(int a_li0da, int a_ly0da) {
  /*    !
    !	We use the fftw api 'Guru execution of plans' (See docu fftw3).
    !	The following conditions must be met:
    !	- The array size, strides, etcetera are the same
    !	  (since those are set by the plan).
    !	- The input and output arrays are the same (in-place) or different
    !	  (out-of-place) if the plan was originally created to be in-place
    !	  or out-of-place, respectively.
    !	- The alignment of the new input/output arrays is the same as that
    !	  of the input/output arrays when the plan was created, unless the
    !	  plan was created with the FFTW_UNALIGNED flag.
    !
    ! We use out-of-place execution.
    ! Don't use the same temp array for input and output!!!
    !
    Complex,dimension(:,:),allocatable:: tmpin, tmpout, cmplxtmp
    Real,dimension(:,:),allocatable:: realtmp
  */
  /*const int fft_dims[]={a_ly0da};
    const int fft_dims_red[]={a_ly0da/2+1};*/
  hipError_t cuda_err;
  int iStream;
  long int mem_free, mem_total;


  li0da=a_li0da;
  ly0da=a_ly0da;
  /*printf("Initializing fourier_cufft with li0da=%u and ly0da=%u.\n",li0da,ly0da);*/

  facnny=1.0/(double)ly0da;

  /* Allocate data storage on the device */

  /* next array is needed for the block oriented FFT */
  /*printf("Allocated so far: %lu. Trying to allocate %u bytes on the device.\n",allocatedDeviceMemory,
    2*lbg0*li0da*(ly0da/2+1)*sizeof(hipfftDoubleComplex));*/
  /*cuda_err = hipMalloc((void**)&dev_cmplxblock_with_da,
			lbg0*li0da*(ly0da/2+1)*sizeof(hipfftDoubleComplex));
  if (cuda_err!=hipSuccess) {
    printf("Error with memory allocation on the device for dev_cmplxblock_with_da.\n%s\n",
	   hipGetErrorString(cuda_err));
  } else {
    allocatedDeviceMemory += lbg0*li0da*(ly0da/2+1)*sizeof(hipfftDoubleComplex);
  }
  */
  /* How much memory on the GPU does the plans need? */
  //hipfftPlan1d(&c2r_plan_y_block,ly0da,HIPFFT_Z2D,li0da*2*lbg0);

  dp_temp_with_da = (hipfftDoubleComplex **)malloc(nStreams*sizeof(hipfftDoubleComplex*));
  c2r_plan_y_block = (hipfftHandle*) malloc(nStreams*sizeof(hipfftHandle));
  r2c_plan_y_block = (hipfftHandle*) malloc(nStreams*sizeof(hipfftHandle));
  for (iStream=0;iStream<nStreams;iStream++) {
    hipMemGetInfo((size_t*)&mem_free, (size_t*)&mem_total);
    /*printf("Allocating %lu bytes in stream %u of %u...., free is %lu of %lu\nallocated so far: %lu\n",
	   2*lbg0/nParts*li0da*(ly0da/2+1)*sizeof(hipfftDoubleComplex),
	   iStream,nStreams,mem_free,mem_total,allocatedDeviceMemory);*/
    cuda_err = hipMalloc((void**)&dp_temp_with_da[iStream],
			  2*lbg0*li0da*(ly0da/2+1)*sizeof(hipfftDoubleComplex)/nParts);
    if (cuda_err != hipSuccess) {
      printf("hipMalloc for dp_temp_with_da[%u] and size of %u gave an error:\n%s",
	     iStream,2*lbg0*li0da*(ly0da/2+1)*sizeof(hipfftDoubleComplex)/nParts,
	     hipGetErrorString(cuda_err));
    } else {
      allocatedDeviceMemory += 2*lbg0*li0da*(ly0da/2+1)*sizeof(hipfftDoubleComplex)/nParts;
    }
    //printf("successful.\n");
    //printf("Setting plan for 1D FFT with dim=%u, howmany=%u.\n",ly0da,li0da*2*lbg0/nParts);
    hipfftPlan1d(&c2r_plan_y_block[iStream],ly0da,HIPFFT_Z2D,li0da*2*lbg0/nParts);
    hipfftSetStream(c2r_plan_y_block[iStream],mystreams[iStream]->cudaStream);
    hipfftPlan1d(&r2c_plan_y_block[iStream],ly0da,HIPFFT_D2Z,li0da*lbg0/nParts);
    hipfftSetStream(r2c_plan_y_block[iStream],mystreams[iStream]->cudaStream);
  }

  /*hipfftPlan1d(&r2c_plan_y_block,ly0da,HIPFFT_D2Z,li0da*lbg0);*/

}

#if 0
/* Copies the inarr to a larger array (larger in width nj0 ->ly0da/2+1).
   Rows 0->nj0 are copied, nj0+1->ly0da/2+1 are set to zero.
   One threadblock per x-y plane and one thread for each point in the x-y plane
   of the larger output array.
   PROBLEM: This easily exceeds 1024 threads per block.
   Therefore the xy-plane is tiled into nTiles, given in gridDim.x*/
__global__ void dev_copy_and_zero_for_dealiasing(hipfftDoubleComplex *inarr, int iwidth, int iheight,
						 hipfftDoubleComplex *outarr,int owidth, int oheight) {
  /* double _Complex inarr[howmany][iheight][iwidth];
     double _Complex outarr[howmany][oheight][owidth];
     gridDim.x = nTiles; gridDim.y = howmany;
     blockDim.x = owidth; blockDim.y=oheight; 
  */
  int nLinesPerTile = oheight/gridDim.x;
  int iOffset = blockIdx.y*iwidth*iheight + blockIdx.x*nLinesPerTile*iwidth 
    + threadIdx.y*iwidth;
  int ilind;
  int olind = blockIdx.y*owidth*oheight + blockIdx.x*nLinesPerTile*owidth
    + threadIdx.y*owidth+threadIdx.x;

  if (threadIdx.x < iwidth) {
    /* just copy */
    ilind = iOffset+threadIdx.x;
    outarr[olind] = inarr[ilind];
  } else {
    /* set entry to zero, x is real part, y is imaginary part. */
    outarr[olind].x = 0.0;
    outarr[olind].y = 0.0;
  }
}

/* Wrapper for the cuda function dev_copy_and_zero_for_dealiasing. */
void copy_and_zero_for_dealiasing(hipfftDoubleComplex *inarr, int iwidth, int iheight,
				  hipfftDoubleComplex *outarr,int owidth, int oheight,
				  int howmany, int streamId) {
  int nXYPoints,nLinesPerTile,nTiles;
  dim3 dimBlock;
  dim3 dimGrid;
  hipError_t cuda_err;  
  
  /*printf("inarr: w=%u, h=%u, out: w=%u, h=%u, howmany = %u\n",iwidth,iheight,
    owidth,oheight,howmany);*/

  if (iheight != oheight) 
    printf("\n----- iheight MUST EQUAL oheight in copy_and_zero_for_dealiasing -----\n\n");

  nXYPoints = owidth*oheight;
  if (nXYPoints<=maxThreadsPerBlock) {
    dimBlock.x=owidth;
    dimBlock.y=oheight;
    dimGrid.x = 1;
    dimGrid.y=howmany;
    dev_copy_and_zero_for_dealiasing<<<dimGrid,dimBlock,
      0,mystreams[streamId]->cudaStream>>>(inarr,iwidth,iheight,
					   dp_temp_with_da[streamId],owidth,oheight);
  } else {
    /* separate the height in several tiles */
    nLinesPerTile = maxThreadsPerBlock/owidth;
    nTiles = oheight/nLinesPerTile;
    if (oheight%nLinesPerTile) nTiles++;
    
    if (oheight%nTiles) {
      printf("oheight (%u) must be divisable by nTiles (%u)\n",oheight,nTiles);
    }
    //printf("nLinesPerTile = %u, nTiles = %u\n",nLinesPerTile,nTiles);
    dimBlock.x = owidth;
    dimBlock.y = oheight/nTiles;
    dimGrid.x  = nTiles;
    dimGrid.y  = howmany;
    dev_copy_and_zero_for_dealiasing<<<dimGrid,dimBlock,
      0,mystreams[streamId]->cudaStream>>>(inarr,iwidth,iheight,
					   dp_temp_with_da[streamId],owidth,oheight);
  }
  cuda_err = hipGetLastError();
  if (cuda_err != hipSuccess) printf("copy_and_zero: %s\n",hipGetErrorString(cuda_err));

}

/* transform 2-dimensional array inarr(ky,x) to outarr(y,x)
   ! note : y is the first coordinate */
void to_real_y_hp_only_on_device(hipfftDoubleComplex *inarr, double *outarr, int howmany, int streamId) {
  /*
    ASSUMPTION:
    inarr[howmany][li0da][nj0] and outarr[howmany][li0da][ly0da]
  */
  /*double _Complex temparr[li0da][ly0da/2+1];*/
  //dim3 dimBlock;
  //dim3 dimGrid;
  //hipfftDoubleComplex *temparrblock;
  hipError_t cuda_err;
  hipfftResult cufft_err;
  int klmn,offset,i,j;
  hipfftDoubleComplex *hp_temp_with_da;

  /* for dealiasing in y direction, we have to copy the inarr
     into the starting part of temparr, which is a larger
     array. The remainder of temparr is filled with zeros. */
  /*dimBlock.x=ly0da/2+1;
  dimBlock.y=li0da;
  dimGrid.x=howmany;*/
  /*printf("Calling dev_copy_and_zero_for_dealiasing with grid(%u,%u) and block(%u,%u,%u)\n",
    dimGrid.x,dimGrid.y,dimBlock.x,dimBlock.y,dimBlock.z);*/
  /*dev_copy_and_zero_for_dealiasing<<<dimGrid,dimBlock,
    0,mystreams[streamId]->cudaStream>>>(inarr,nj0,
					 li0da,dp_temp_with_da[streamId],
					 ly0da/2+1,li0da,howmany);*/
  copy_and_zero_for_dealiasing(inarr,nj0,li0da,
			       dp_temp_with_da[streamId],ly0da/2+1,li0da,
			       howmany, streamId);
  /*cuda_err = hipStreamSynchronize(stream[streamId]);*/

#if 0
  /* DEBUG */
  hp_temp_with_da = (hipfftDoubleComplex*)malloc(howmany*li0da*(ly0da/2+1)*sizeof(hipfftDoubleComplex));
  hipMemcpy(hp_temp_with_da,dp_temp_with_da[streamId],
	     howmany*li0da*(ly0da/2+1)*sizeof(hipfftDoubleComplex),hipMemcpyDeviceToHost);

  klmn= 0;
  offset = klmn*li0da*(ly0da/2+1);
  for (i=0;i<li0da;i++) {
    for (j=0;j<ly0da/2+1;j++) {
      printf("(%10.3e %10.3e) ",hp_temp_with_da[offset+i*(ly0da/2+1)+j].x,
	     hp_temp_with_da[offset+i*(ly0da/2+1)+j].y);
    }
    printf("\n");
  }
  printf("\n");
  /*for (klmn=0;klmn<howmany;klmn++) {
    printf("klmn= %u: sum_cabs = %17.6e\n",klmn,sum_cabs(&hp_temp_with_da[klmn*li0da*(ly0da/2+1)],
							 li0da*(ly0da/2+1)));
							 }*/
  free(hp_temp_with_da);
  /* END DEBUG */
#endif

  cufft_err = hipfftExecZ2D(c2r_plan_y_block[streamId], 
			   (hipfftDoubleComplex *)dp_temp_with_da[streamId], 
			   (hipfftDoubleReal *)outarr);
  if (cufft_err != HIPFFT_SUCCESS) printf("Error with hipfftExecZ2D. error_code = %d\n",cufft_err);
  //cuda_err = hipStreamSynchronize(stream[streamId]);
  //if (cuda_err != hipSuccess) printf("hipfftExecZ2D: %s\n",hipGetErrorString(cuda_err));
}
#endif

/* transform 2-dimensional array inarr(ky,x) to outarr(y,x)
   ! note : y is the first coordinate */
void to_real_y_only_on_device(hipfftDoubleComplex *inarr, double *outarr, int howmany, int streamId) {
  /*
    ASSUMPTION:
    inarr[howmany][li0da][ly0da/2+1] and outarr[howmany][li0da][ly0da]
  */
  /*double _Complex temparr[li0da][ly0da/2+1];*/
  //dim3 dimBlock;
  //dim3 dimGrid;
  //hipfftDoubleComplex *temparrblock;
  //hipError_t cuda_err;
  hipfftResult cufft_err;

  cufft_err = hipfftExecZ2D(c2r_plan_y_block[streamId], 
			   (hipfftDoubleComplex *)inarr, 
			   (hipfftDoubleReal *)outarr);
  if (cufft_err != HIPFFT_SUCCESS) printf("Error with hipfftExecZ2D. error_code = %d\n",cufft_err);
  //cuda_err = hipStreamSynchronize(stream[streamId]);
  //if (cuda_err != hipSuccess) printf("hipfftExecZ2D: %s\n",hipGetErrorString(cuda_err));
}

/* just a small kernel which copies the first nj0 entries of the first
   parameter to the second one.
   
   gridDim.x=lbg0
   gridDim.y=nTiles
   blockDim.x=nj0
   blockDim.y=li0da/nTiles
*/
__global__ void copy_only_nondealiased(hipfftDoubleComplex *dev_cmplxblock_with_da,int iwidth,int iheight,
				       hipfftDoubleComplex *dev_outarr, int owidth,double facnny) {

  int o_index = blockIdx.x*owidth*iheight + blockIdx.y*blockDim.y*owidth + threadIdx.y*owidth + threadIdx.x; 
  int i_index = blockIdx.x*iwidth*iheight + blockIdx.y*blockDim.y*iwidth + threadIdx.y*iwidth + threadIdx.x; 
  dev_outarr[o_index].x = dev_cmplxblock_with_da[i_index].x*facnny;
  dev_outarr[o_index].y = dev_cmplxblock_with_da[i_index].y*facnny;
}

/* We assume oheight=iheight. */
void copy_only_nondealiased_wrapper(hipfftDoubleComplex *dev_cmplxblock_with_da,int iwidth, int iheight,
				    hipfftDoubleComplex *dev_outarr, int owidth, double facnny, int iStream) {
  dim3 grid,threadblock; //(lbg0,1);
  //dim3 threadblock(li0da,nj0,1);
  hipError_t cuda_err;
  int dimXYPlane, nTiles; 
  
  dimXYPlane = owidth*iheight;
  nTiles = (dimXYPlane+1023)/1024;

  while ((iheight%nTiles !=0)&&(nTiles<128)) nTiles++;
  if (iheight%nTiles !=0) {
    printf("iheight = %u, cannot be tiled in copy_only_nondealiased_wrapper! Aborting!\n");
    exit(1);
  }/* else {
    printf("We are using %u tiles in copy_only_dealiased.\n",nTiles);
    }*/
  grid.x = lbg0/nParts;
  grid.y = nTiles;
  grid.z = 1;
  threadblock.x=owidth;
  threadblock.y=iheight/nTiles;
  threadblock.z=1;
  copy_only_nondealiased<<<grid,threadblock,0,mystreams[iStream]->cudaStream>>>
    (dev_cmplxblock_with_da,iwidth,iheight,dev_outarr,owidth,facnny);
  cuda_err = hipGetLastError();
  if (cuda_err != hipSuccess) {
    printf("Error with kernel launch.(copy_only_nondealiased)\n");
    printf("%s\n",hipGetErrorString(cuda_err));
  }
				    }
void to_fourier_y_hp_only_on_device(double *inarr, hipfftDoubleComplex *dev_outarr, int iStream) {
  /*
    Real,    Dimension(0:ly0da-1,0:li0da/n_procs_y-1), intent(in)   :: inarr
    Complex, Dimension(0:nj0-1, 0:li0da/n_procs_y-1)  , intent(out) :: outarr
    Complex, Dimension(0:ly0da/2,0:li0da/n_procs_y-1)              :: temparr
    ASSUMPTION:
    double inarr[lbg0][li0da][ly0da];
    double _Complex outarr[lbg0][li0da][nj0];
  */
  //double _Complex temparr[lbg0][li0da][ly0da/2+1];

  /*hipfftExecD2Z(r2c_plan_y_block[iStream], (hipfftDoubleReal *)inarr, 
    (hipfftDoubleComplex *)dev_cmplxblock_with_da);*/
#if 0
  double _Complex *tmp_arr;
  int j,klmn;
#endif

  hipfftExecD2Z(r2c_plan_y_block[iStream], (hipfftDoubleReal *)inarr, 
	       (hipfftDoubleComplex *)(mystreams[iStream]->dp_fordeal));
#if 0
  tmp_arr = (double _Complex*)malloc(li0da*(ly0da/2+1)*lbg0/nParts*sizeof(hipfftDoubleComplex));
  hipMemcpy(tmp_arr,mystreams[iStream]->dp_fordeal,
	     li0da*(ly0da/2+1)*lbg0/nParts*sizeof(double _Complex),
	     hipMemcpyDeviceToHost);
#if 0
  for (klmn=0;klmn<lbg0/nParts;klmn++) {
    printf("klmn=%u, %f\n",klmn, sum_cabs(tmp_arr+klmn*li0da*(ly0da/2+1),li0da*(ly0da/2+1)));
  }

  /*for (j=0;j<ly0da/2+1;j++) {
    printf("(%f %f) ",creal(tmp_arr[j]),cimag(tmp_arr[j]));
  }
  printf("\n");*/
#endif
  printf("after FFT, before dealiasing (y: 1->ly0da/2+1) is %f\n",
	 sum_cabs((hipfftDoubleComplex*)tmp_arr,li0da*(ly0da/2+1)*lbg0/nParts));
  free(tmp_arr);
#endif

  copy_only_nondealiased_wrapper((hipfftDoubleComplex *)(mystreams[iStream]->dp_fordeal),
				 ly0da/2+1,li0da,
				 dev_outarr,nj0,facnny,iStream);
}

extern "C" void finalize_fourier_cufft() {
  int iStream;
  /* free the GPU memory */
  //hipFree(dev_cmplxarr);
  //hipFree(dev_cmplxblock_with_da);

  /* free the FFT plans */
  for (iStream=0;iStream<nStreams;iStream++) {
    hipfftDestroy(c2r_plan_y_block[iStream]);
    hipFree(dp_temp_with_da[iStream]);
    hipfftDestroy(r2c_plan_y_block[iStream]);
  }
  free(dp_temp_with_da);
  free(c2r_plan_y_block);
  free(r2c_plan_y_block);
}
