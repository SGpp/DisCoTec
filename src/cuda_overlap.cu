#include <stdio.h>
#include "cuda_overlap.h"

extern unsigned long int allocatedDeviceMemory;

Streamdata::Streamdata(int li0,int lj0,int ly0da, int nXYPlanes) {
  hipError_t cuda_err;
  int dimXYPlane;
  int nXYPlanesPerPart;

  dimXYPlane = li0*lj0;
  nXYPlanesPerPart = nXYPlanes/nParts;

  /*printf("Allocating dp_data with %lu bytes.\n",nXYPlanesPerPart*dimXYPlane*sizeof(double _Complex));*/
  cuda_err = hipMalloc((void**)&dp_data,nXYPlanesPerPart*dimXYPlane*sizeof(double _Complex));
  if (cuda_err != hipSuccess) {
    printf("overlap hipMalloc 0: %s\n",hipGetErrorString(cuda_err));
  } else {
    allocatedDeviceMemory += nXYPlanesPerPart*dimXYPlane*sizeof(double _Complex);
  }

  /*printf("Allocating dp_temp with %lu bytes.\n",nXYPlanesPerPart*li0*(ly0da/2+1)*sizeof(double _Complex));*/
  cuda_err = hipMalloc((void**)&dp_temp,nXYPlanesPerPart*li0*(ly0da/2+1)*sizeof(double _Complex));
  if (cuda_err != hipSuccess) {
    printf("overlap hipMalloc 1: %s\n",hipGetErrorString(cuda_err));
  } else {
    allocatedDeviceMemory += nXYPlanesPerPart*li0*(ly0da/2+1)*sizeof(double _Complex);
  }

  /*printf("Allocating dp_data with %lu bytes.\n",nXYPlanesPerPart*li0*(ly0da/2+1)*sizeof(double _Complex));*/
  cuda_err = hipMalloc((void**)&dp_fordeal,nXYPlanesPerPart*li0*(ly0da/2+1)*sizeof(double _Complex));
  if (cuda_err != hipSuccess) {
    printf("overlap hipMalloc 2: %s\n",hipGetErrorString(cuda_err));
  } else {
    allocatedDeviceMemory += nXYPlanesPerPart*li0*(ly0da/2+1)*sizeof(double _Complex);
  }

  //hipStreamCreateWithFlags(&cudaStream,hipStreamNonBlocking);
  hipStreamCreate(&cudaStream);

}

Streamdata::~Streamdata() {
  hipStreamDestroy(cudaStream);

  hipFree(dp_data);
  hipFree(dp_temp);
  hipFree(dp_fordeal);
}
